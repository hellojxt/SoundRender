#include "hip/hip_runtime.h"
#include "gui.h"
#include "window.h"
#include "objIO.h"
#include "audio_window.h"
#include "portaudio.h"
#include <math.h>
using namespace SoundRender;


#define SAMPLE_RATE   (44100)
#define FRAMES_PER_BUFFER  (64)
#ifndef M_PI
#define M_PI  (3.14159265)
#endif
#define TABLE_SIZE (200)

//test mode on: play sine wave while the GUI window is open
//test mode off: work as the sound renderer
// #define test

#ifdef test
typedef struct
{
    float sine[TABLE_SIZE];
    int left_phase;
    int right_phase;
}
paSoundData;

#else
struct paSoundData
{
    AudioWapper* aw;

    paSoundData(AudioWapper* another) :aw(another) {};
};
#endif

static int patestCallback(const void* inputBuffer, void* outputBuffer,
    unsigned long framesPerBuffer,
    const PaStreamCallbackTimeInfo* timeInfo,
    PaStreamCallbackFlags statusFlags,
    void* userData)
{
    paSoundData* data = (paSoundData*)userData;
    float* out = (float*)outputBuffer;
    unsigned long i;

    (void) timeInfo; /* Prevent unused variable warnings. */
    (void) statusFlags;
    (void) inputBuffer;

    for (i = 0; i < framesPerBuffer; i++)
    {
#ifdef test
        *out++ = data->sine[data->left_phase];  /* left */
        *out++ = data->sine[data->right_phase];  /* right */
        data->left_phase += 1;
        if (data->left_phase >= TABLE_SIZE) data->left_phase -= TABLE_SIZE;
        data->right_phase += 1;
        if (data->right_phase >= TABLE_SIZE) data->right_phase -= TABLE_SIZE;
#else
        float sound = AudioWapper::CallbackForSound(data->aw);
        *out++ = sound;  /* left */
        *out++ = sound;  /* right */
#endif

    }

    return paContinue;
}



int main()
{
    auto filename = std::string(ASSET_DIR) + std::string("/meshes/bunny.obj");
    auto mesh = loadOBJ(filename, true);
    GUI gui;
    MeshRender render;
    AudioWindow audio_window;
    render.load_mesh(mesh.vertices, mesh.triangles);
    audio_window.link_mesh_render(&render);
    gui.add_window(&render);
    gui.add_window(&audio_window);


    //==========portaudio====================
    PaStreamParameters outputParameters;
    PaStream* stream;
    PaError err;

   


#ifdef test
    paSoundData data;
    int i;

    for (i = 0; i < TABLE_SIZE; i++)
    {
        data.sine[i] = sin(((double)i / TABLE_SIZE) * M_PI * 2.0 ) ;
    }
    data.left_phase = data.right_phase = 0;
#else

    paSoundData data=paSoundData(&audio_window.audio);
#endif
  

    err=Pa_Initialize();
    if (err != paNoError) goto error;

    outputParameters.device = Pa_GetDefaultOutputDevice(); /* default output device */
    outputParameters.channelCount = 2;       /* stereo output */
    outputParameters.sampleFormat = paFloat32; /* 32 bit floating point output */
    outputParameters.suggestedLatency = Pa_GetDeviceInfo(outputParameters.device)->defaultLowOutputLatency;
    outputParameters.hostApiSpecificStreamInfo = NULL;

    err = Pa_OpenStream(
        &stream,
        NULL, /* no input */
        &outputParameters,
        SAMPLE_RATE,
        FRAMES_PER_BUFFER,
        paClipOff,      /* we won't output out of range samples so don't bother clipping them */
        patestCallback,
        &data);
    if (err != paNoError) goto error;
   
    err = Pa_StartStream(stream);
    if (err != paNoError) goto error;
    //==========portaudio====================
    
    gui.start();


    //==========portaudio====================
    err = Pa_StopStream(stream);
    if (err != paNoError) goto error;
    err = Pa_CloseStream(stream);
    if (err != paNoError) goto error;
    err = Pa_Terminate();
    if (err != paNoError) goto error;
    //==========portaudio====================

    return 0;

error:
    Pa_Terminate();
    fprintf(stderr, "An error occurred while using the portaudio stream\n");
    fprintf(stderr, "Error number: %d\n", err);
    fprintf(stderr, "Error message: %s\n", Pa_GetErrorText(err));
    return 0;
}













