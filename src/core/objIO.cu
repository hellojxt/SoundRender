#include "hip/hip_runtime.h"
#include "objIO.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

namespace SoundRender
{

	Mesh::Mesh(CArr<float3> vertices_, CArr<int3> triangles_)
	{
		vertices = vertices_;
		triangles = triangles_;
	}

    Mesh::Mesh(CArr<float3> vertices_, CArr<int3> triangles_, CArr<float3> tex_, CArr<int3> tex_triangles_)
	{
		vertices = vertices_;
		triangles = triangles_;
		vertex_texcoords = tex_;
		tex_triangles = tex_triangles_;
	}

	void Mesh::print()
	{
		std::cout << "Vertices:\n";
		for (auto v : vertices.m_data)
			std::cout << "(" << v.x << "," << v.y << "," << v.z << ")\n";
		std::cout << "Triangles:\n";
		for (auto f : triangles.m_data)
			std::cout << "[" << f.x << "," << f.y << "," << f.z << "]\n";
		if(!vertex_texcoords.isEmpty())
		{
			std::cout << "Textures:\n";
			for (auto f : vertex_texcoords.m_data)
				std::cout << "[" << f.x << "," << f.y << "]\n";
		}
	}

	void Mesh::writeOBJ(std::string filename)
	{
	}

	Mesh loadOBJ(std::string file_name, bool log)
	{
		// std::ofstream fout("/home/jiaming/Self/output/temp2.txt", std::ios::trunc);
		CArr<float3> vertices;
		CArr<int3> triangles;
		CArr<int3> tex_triangles;
		CArr<float3> vertex_texcoords;
		std::stringstream ss;
		std::ifstream in_file(file_name);
		std::string line = "";
		std::string prefix = "";

		// std::cout << "Start reading\n";
		// File open error check
		if (!in_file.is_open())
		{
			std::cout << "Error opening file: " << file_name << "\n";
			exit(1);
		}

		// Read one line at a time
		while (std::getline(in_file, line))
		{
			// Get the prefix of the line
			ss.clear();
			ss.str(line);
			ss >> prefix;

			if (prefix == "#")
			{
			}
			else if (prefix == "o")
			{
			}
			else if (prefix == "s")
			{
			}
			else if (prefix == "use_mtl")
			{
			}
			else if (prefix == "v") // Vertex position
			{
				float3 tmp;
				ss >> tmp.x >> tmp.y >> tmp.z;
				vertices.pushBack(tmp);
				// fout << "v " << tmp.x << " " << tmp.y << " " << tmp.z << "\n";
			}
			else if (prefix == "vt")
			{
				float3 temp_vec2;
				ss >> temp_vec2.x >> temp_vec2.y;
				temp_vec2.z = 0.0f;
				vertex_texcoords.pushBack(temp_vec2);
				// fout << "vt " << temp_vec2.x << " " << temp_vec2.y << "\n";
			}
			else if (prefix == "vn")
			{
				// here we omit norm_inds.
				// ss >> temp_vec3.x >> temp_vec3.y >> temp_vec3.z;
				// vertex_normals.push_back(temp_vec3);
			}
			else if (prefix == "f")
			{
				int3 vert_inds;
				int3 norm_inds;
				int3 text_inds;
				char slash;

				ss >> vert_inds.x;
				if(ss.peek() == '/')
				{
					ss.get();
					if(ss.peek() == '/') // v1//n1 v2//n2 v3//n3
					{
						ss >> slash >> norm_inds.x >> vert_inds.y >> slash >> slash >> norm_inds.y
							>> vert_inds.z >> slash >> slash >> norm_inds.z;
						triangles.pushBack(vert_inds);
						// here we omit norm_inds.
					}
					else{
						ss >> text_inds.x;
						if(ss.peek() == '/') // v1/t1/n1 v2/t2/n2 v3/t3/n3
						{
							ss >> slash >> norm_inds.x >> vert_inds.y >> slash >> text_inds.y >> slash
								>> norm_inds.y >> vert_inds.z >> slash >> text_inds.z >> slash >> norm_inds.z;
							triangles.pushBack(vert_inds);
							tex_triangles.pushBack(text_inds);
							// fout << "f " << vert_inds.x << "/" << text_inds.x << " " << vert_inds.y << "/" << text_inds.y
							// 	<< " " << vert_inds.z << "/" << text_inds.z << "\n";
							// fout << "f " << vert_inds.x << " " << vert_inds.y << " " << vert_inds.z << "\n";
							// here we omit norm_inds.
						}
						else{ // v1/t1 v2/t2 v3/t3
							ss >> vert_inds.y >> slash >> text_inds.y >> vert_inds.z >> slash >> text_inds.z;
							triangles.pushBack(vert_inds);
							tex_triangles.pushBack(text_inds);
						}
					}
				}
				else{ // v1 v2 v3
					ss >> vert_inds.y >> vert_inds.z;
					triangles.pushBack(vert_inds);
				}
			}
		}
		std::for_each(triangles.begin(), triangles.end(), [](int3& a){ --a.x; --a.y; --a.z; });
		std::for_each(tex_triangles.begin(), tex_triangles.end(), [](int3& a){ --a.x; --a.y; --a.z; });
		if (log)
		{
			// LOG
			std::cout << "Vertices number: " << vertices.size() << "\n";
			std::cout << "Triangles number: " << triangles.size() << "\n";
			std::cout << "TextureVert number: " << vertex_texcoords.size() << "\n";
			std::cout << "TextureTri number : " << tex_triangles.size() << "\n";
			// Loaded success
			std::cout << "OBJ file:" << file_name << " loaded!"
					  << "\n";
		}
		if(vertex_texcoords.isEmpty())
			return Mesh(vertices, triangles);
		return Mesh(vertices, triangles, vertex_texcoords, tex_triangles);
	}

}
