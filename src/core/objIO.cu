#include "hip/hip_runtime.h"
#include "objIO.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

namespace SoundRender
{

	Mesh::Mesh(CArr<float3> vertices_, CArr<int3> triangles_)
	{
		vertices = vertices_;
		triangles = triangles_;
	}
	void Mesh::print()
	{
		std::cout << "Vertices:\n";
		for (auto v : vertices.m_data)
			std::cout << "(" << v.x << "," << v.y << "," << v.z << ")\n";
		std::cout << "Triangles:\n";
		for (auto f : triangles.m_data)
			std::cout << "[" << f.x << "," << f.y << "," << f.z << "]\n";
	}

	void Mesh::writeOBJ(std::string filename)
	{
	}

	Mesh loadOBJ(std::string file_name, bool log)
	{
		CArr<float3> vertices;
		CArr<int3> triangles;
		std::stringstream ss;
		std::ifstream in_file(file_name);
		std::string line = "";
		std::string prefix = "";

		// std::cout << "Start reading\n";
		// File open error check
		if (!in_file.is_open())
		{
			std::cout << "Error opening file: " << file_name << "\n";
			exit(1);
		}

		// Read one line at a time
		while (std::getline(in_file, line))
		{
			// Get the prefix of the line
			ss.clear();
			ss.str(line);
			ss >> prefix;

			if (prefix == "#")
			{
			}
			else if (prefix == "o")
			{
			}
			else if (prefix == "s")
			{
			}
			else if (prefix == "use_mtl")
			{
			}
			else if (prefix == "v") // Vertex position
			{
				float3 tmp;
				ss >> tmp.x >> tmp.y >> tmp.z;
				vertices.pushBack(tmp);
			}
			else if (prefix == "vt")
			{
				// ss >> temp_vec2.x >> temp_vec2.y;
				// vertex_texcoords.push_back(temp_vec2);
			}
			else if (prefix == "vn")
			{
				// ss >> temp_vec3.x >> temp_vec3.y >> temp_vec3.z;
				// vertex_normals.push_back(temp_vec3);
			}
			else if (prefix == "f")
			{
				int tmp;
				int counter = 0;
				std::vector<int> tmp_inds;
				while (ss >> tmp)
				{
					// Pushing indices into correct arrays
					if (counter == 0)
						tmp_inds.push_back(tmp - 1);
					// else if (counter == 1)
					// 	vertex_texcoord_indicies.push_back(temp_glint);
					// else if (counter == 2)
					// 	vertex_normal_indicies.push_back(temp_glint);

					// Handling characters
					if (ss.peek() == '/')
					{
						++counter;
						ss.ignore(1, '/');
					}
					else if (ss.peek() == ' ')
					{
						counter = 0;
						ss.ignore(1, ' ');
					}

					// Reset the counter
					if (counter > 2)
						counter = 0;
				}
				triangles.pushBack(make_int3(tmp_inds[0], tmp_inds[1], tmp_inds[2]));
			}
		}
		if (log)
		{
			// LOG
			std::cout << "Vertices number: " << vertices.size() << "\n";
			std::cout << "Triangles number: " << triangles.size() << "\n";
			// Loaded success
			std::cout << "OBJ file:" << file_name << " loaded!"
					  << "\n";
		}

		return Mesh(vertices, triangles);
	}

}
