#include "hip/hip_runtime.h"
#include "objIO.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

namespace SoundRender
{

	Mesh::Mesh(CArr<float3> vertices_, CArr<int3> triangles_)
	{
		vertices = vertices_;
		triangles = triangles_;
	}

    Mesh::Mesh(CArr<float3> vertices_, CArr<int3> triangles_, CArr<float3> tex_, CArr<int3> tex_triangles_)
	{
		vertices = vertices_;
		triangles = triangles_;
		vertex_texcoords = tex_;
		tex_triangles = tex_triangles_;
	}

	void Mesh::print()
	{
		std::cout << "Vertices:\n";
		for (auto v : vertices.m_data)
			std::cout << "(" << v.x << "," << v.y << "," << v.z << ")\n";
		std::cout << "Triangles:\n";
		for (auto f : triangles.m_data)
			std::cout << "[" << f.x << "," << f.y << "," << f.z << "]\n";
		if(!vertex_texcoords.isEmpty())
		{
			std::cout << "Textures:\n";
			for (auto f : vertex_texcoords.m_data)
				std::cout << "[" << f.x << "," << f.y << "]\n";
		}
	}

	void Mesh::writeOBJ(std::string filename)
	{
	}

    void Mesh::loadMaterial(const std::filesystem::path& fileName, const std::string& materialName)
    {
        std::string prefix;
        std::ifstream fin(fileName.c_str());
        if(!fin.is_open())
            std::cout << "Fail to open mtl file : " << fileName.c_str() << "\n";
        std::string currMtlName;
        while(fin >> prefix)
        {
            if(prefix == "newmtl")
            {
                fin >> currMtlName;
                if(currMtlName == materialName)
                    break;
            }
        }
        while (true)
        {
            fin >> prefix;
			if(prefix == "newmtl" || !fin.good())
                break;
            else if(prefix == "Ns")
            {
                fin >> specularExp;
            }
            else if(prefix == "Ka")
            {
                fin >> ambientCoeff.x >> ambientCoeff.y >> ambientCoeff.z;
            }
            else if(prefix == "Kd")
            {
                fin >> diffuseCoeff.x >> diffuseCoeff.y >> diffuseCoeff.z;
            }
            else if(prefix == "Ks")
            {
                fin >> specularCoeff.x >> specularCoeff.y >> specularCoeff.z;
            }
            else if(prefix == "d")
            {
                fin >> alpha;
            }
			else if(prefix == "map_Kd")
			{
				fin >> texturePicName;
			}
        }
        return;
    }


	Mesh loadOBJ(std::string file_name, bool log)
	{
		std::filesystem::path assetPath{ASSET_DIR};
		#ifdef _WIN32
			assetPath /= L"materials";
		#else
			assetPath /= "materials";
		#endif

		CArr<float3> vertices;
		CArr<int3> triangles;
		CArr<int3> tex_triangles;
		CArr<float3> vertex_texcoords;
		std::stringstream ss;
		std::ifstream in_file(file_name);
		std::string line = "";
		std::string prefix = "";

		// std::cout << "Start reading\n";
		// File open error check
		if (!in_file.is_open())
		{
			std::cout << "Error opening file: " << file_name << "\n";
			exit(1);
		}

		std::string mtlLibName;
		std::string materialName;
		// Read one line at a time
		while (std::getline(in_file, line))
		{
			// Get the prefix of the line
			ss.clear();
			ss.str(line);
			ss >> prefix;

			if (prefix == "#")
			{
			}
			else if (prefix == "o")
			{
			}
			else if (prefix == "s")
			{
			}
			else if (prefix == "usemtl")
			{
				ss >> materialName;
			}
			else if(prefix == "mtllib")
			{
				ss >> mtlLibName;
			}
			else if (prefix == "v") // Vertex position
			{
				float3 tmp;
				ss >> tmp.x >> tmp.y >> tmp.z;
				vertices.pushBack(tmp);
				// fout << "v " << tmp.x << " " << tmp.y << " " << tmp.z << "\n";
			}
			else if (prefix == "vt")
			{
				float3 temp_vec2;
				ss >> temp_vec2.x >> temp_vec2.y;
				temp_vec2.z = 0.0f;
				vertex_texcoords.pushBack(temp_vec2);
				// fout << "vt " << temp_vec2.x << " " << temp_vec2.y << "\n";
			}
			else if (prefix == "vn")
			{
				// here we omit norm_inds.
				// ss >> temp_vec3.x >> temp_vec3.y >> temp_vec3.z;
				// vertex_normals.push_back(temp_vec3);
			}
			else if (prefix == "f")
			{
				int3 vert_inds;
				int3 norm_inds;
				int3 text_inds;
				char slash;

				ss >> vert_inds.x;
				if(ss.peek() == '/')
				{
					ss.get();
					if(ss.peek() == '/') // v1//n1 v2//n2 v3//n3
					{
						ss >> slash >> norm_inds.x >> vert_inds.y >> slash >> slash >> norm_inds.y
							>> vert_inds.z >> slash >> slash >> norm_inds.z;
						triangles.pushBack(vert_inds);
						// here we omit norm_inds.
					}
					else{
						ss >> text_inds.x;
						if(ss.peek() == '/') // v1/t1/n1 v2/t2/n2 v3/t3/n3
						{
							ss >> slash >> norm_inds.x >> vert_inds.y >> slash >> text_inds.y >> slash
								>> norm_inds.y >> vert_inds.z >> slash >> text_inds.z >> slash >> norm_inds.z;
							triangles.pushBack(vert_inds);
							tex_triangles.pushBack(text_inds);
							// fout << "f " << vert_inds.x << "/" << text_inds.x << " " << vert_inds.y << "/" << text_inds.y
							// 	<< " " << vert_inds.z << "/" << text_inds.z << "\n";
							// fout << "f " << vert_inds.x << " " << vert_inds.y << " " << vert_inds.z << "\n";
							// here we omit norm_inds.
						}
						else{ // v1/t1 v2/t2 v3/t3
							ss >> vert_inds.y >> slash >> text_inds.y >> vert_inds.z >> slash >> text_inds.z;
							triangles.pushBack(vert_inds);
							tex_triangles.pushBack(text_inds);
						}
					}
				}
				else{ // v1 v2 v3
					ss >> vert_inds.y >> vert_inds.z;
					triangles.pushBack(vert_inds);
				}
			}
		}
		std::for_each(triangles.begin(), triangles.end(), [](int3& a){ --a.x; --a.y; --a.z; });
		std::for_each(tex_triangles.begin(), tex_triangles.end(), [](int3& a){ --a.x; --a.y; --a.z; });
		if (log)
		{
			// LOG
			std::cout << "Vertices number: " << vertices.size() << "\n";
			std::cout << "Triangles number: " << triangles.size() << "\n";
			std::cout << "TextureVert number: " << vertex_texcoords.size() << "\n";
			std::cout << "TextureTri number : " << tex_triangles.size() << "\n";
			// Loaded success
			std::cout << "OBJ file:" << file_name << " loaded!"
					  << "\n";
		}
		Mesh mesh = vertex_texcoords.isEmpty()? Mesh(vertices, triangles) : Mesh(vertices, triangles, vertex_texcoords, tex_triangles);
		mesh.loadMaterial(assetPath / mtlLibName, materialName);
		return mesh;
	}

}
