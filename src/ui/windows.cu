#include "hip/hip_runtime.h"
#include "window.h"
#include "gui_kernel.h"
#include "stb_image.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

namespace SoundRender
{
    void MeshRender::loadTexture(const char* path)
    {
        int width, height, channels;
        unsigned char* data = stbi_load(path, &width, &height, &channels, 0);
        if(data == nullptr)
        {
            std::cout << "Texture failed to load at path : " << path << std::endl;
            stbi_image_free(data);
            textureID = 0;
        }
        GLenum format;
        switch (channels)
        {
        case 1:
            format = GL_RED;
            break;
        case 3:
            format = GL_RGB;
            break;
        case 4:
            format = GL_RGBA; 
            break;
        }
        glBindTexture(GL_TEXTURE_2D, textureID);
        glTexImage2D(GL_TEXTURE_2D, 0, format, width, height, 0, format, GL_UNSIGNED_BYTE, data);
        // glGenerateMipmap(GL_TEXTURE_2D);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        stbi_image_free(data);
        glBindTexture(GL_TEXTURE_2D, 0);
        return;
    }

    void MeshRender::loadSkycube(const std::string& path)
    {
        glBindTexture(GL_TEXTURE_CUBE_MAP, skycubeID);
        auto loadOneFacet = [&path](unsigned int MACRO, const char* name){
            int width, height, channels;
            unsigned char* data = stbi_load((path + name).c_str(), &width, &height, &channels, 0);
            if(data == nullptr)
            {
                std::cout << "Texture failed to load at path : " << path << std::endl;
                stbi_image_free(data);
                return;
            }
            GLenum format;
            switch (channels)
            {
            case 1:
                format = GL_RED;
                break;
            case 3:
                format = GL_RGB;
                break;
            case 4:
                format = GL_RGBA; 
                break;
            }
            glTexImage2D(MACRO, 0, format, width, height, 0, format, GL_UNSIGNED_BYTE, data);
            stbi_image_free(data);
            return;
        };

        loadOneFacet(GL_TEXTURE_CUBE_MAP_POSITIVE_X, "cm_pos_x.png");
        loadOneFacet(GL_TEXTURE_CUBE_MAP_NEGATIVE_X, "cm_neg_x.png");
        loadOneFacet(GL_TEXTURE_CUBE_MAP_POSITIVE_Y, "cm_pos_z.png");
        loadOneFacet(GL_TEXTURE_CUBE_MAP_NEGATIVE_Y, "cm_neg_z.png");
        loadOneFacet(GL_TEXTURE_CUBE_MAP_POSITIVE_Z, "cm_pos_y.png");
        loadOneFacet(GL_TEXTURE_CUBE_MAP_NEGATIVE_Z, "cm_neg_y.png");

        glTexParameteri(GL_TEXTURE_CUBE_MAP, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_CUBE_MAP, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_CUBE_MAP, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
        glTexParameteri(GL_TEXTURE_CUBE_MAP, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
        glBindTexture(GL_TEXTURE_CUBE_MAP, 0);
        return;
    }


    void MeshRender::Prepare(std::string mtlLibName)
    {
        mtlLib = mtlLibName;
        return;
    }

    void MeshRender::init()
    {
        glGenTextures(1, &textureID);
        glGenTextures(1, &skycubeID);
        material = loadMaterial(std::string(ASSET_DIR) + std::string("/materials/") + mtlLib, "Ceramic_Glazed");
        bool useTexture = material.texturePicName.length() != 0;
        if(useTexture)
        {
            std::string texturePath = std::string(ASSET_DIR) + std::string("/materials/") + material.texturePicName;
            loadTexture(texturePath.c_str());
        }
        glGenFramebuffers(1, &framebuffer);
        glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
        // glEnable(GL_TEXTURE_2D);
        glGenTextures(1, &textureColorbuffer);
        glBindTexture(GL_TEXTURE_2D, textureColorbuffer);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, 1000, 1000, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, textureColorbuffer, 0);
        glBindTexture(GL_TEXTURE_2D, 0);
        
        glGenRenderbuffers(1, &rbo);
        glBindRenderbuffer(GL_RENDERBUFFER, rbo);
        glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, 1000, 1000);
        glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_STENCIL_ATTACHMENT, GL_RENDERBUFFER, rbo);
        glBindRenderbuffer(GL_RENDERBUFFER, 0);
        if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
            LOG_ERROR("Framebuffer not complete!")
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        shader.load(std::string(SHADER_DIR) + std::string("/mesh.vert"),
                    std::string(SHADER_DIR) + std::string("/mesh.frag"));
        shader.use();
        shader.setInt("Texture", 0);
        shader.setInt("skyCube", 1);
        shader.setVec3("ambientCoeff", material.ambientCoeff.x, material.ambientCoeff.y, material.ambientCoeff.z);
        shader.setVec3("diffuseCoeff", material.diffuseCoeff.x, material.diffuseCoeff.y, material.diffuseCoeff.z);
        shader.setVec3("specularCoeff", material.specularCoeff.x, material.specularCoeff.y, material.specularCoeff.z);
        shader.setFloat("specularExp", material.specularExp);
        shader.setFloat("alpha", material.alpha);
        shader.setInt("useTexture", (int)useTexture);
        shader.setInt("useSkyCube", 0);
    }

    void MeshRender::resize()
    {
        ImVec2 size = ImGui::GetWindowSize();
        if (size.x != wsize.x || size.y != wsize.y)
        {
            wsize = size;
            float tx = wsize.x * OVERSAMPLE, ty = wsize.y * OVERSAMPLE;
            glBindTexture(GL_TEXTURE_2D, textureColorbuffer);
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, tx, ty, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
            glBindTexture(GL_TEXTURE_2D, 0);
            glBindRenderbuffer(GL_RENDERBUFFER, rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, tx, ty);
            glBindRenderbuffer(GL_RENDERBUFFER, 0);
        }
    }

    void MeshRender::updateMesh()
    {
        glGenVertexArrays(1, &meshVAO);
        glGenBuffers(1, &meshVBO);
        glBindVertexArray(meshVAO);
        glBindBuffer(GL_ARRAY_BUFFER, meshVBO);
        glBufferData(GL_ARRAY_BUFFER, sizeof(Triangle) * meshData.size(), meshData.data(), GL_STATIC_DRAW);
        // position attribute
        glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void *)0);
        glEnableVertexAttribArray(0);
        // normal attribute
        glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void *)(3 * sizeof(float)));
        glEnableVertexAttribArray(1);
        // index attribute
        glVertexAttribIPointer(2, 1, GL_INT, 9 * sizeof(float) + sizeof(int), (void *)(9 * sizeof(float)));
        glEnableVertexAttribArray(2);
        // texture attribute
        glVertexAttribPointer(3, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void*)(6 * sizeof(float)));
        glEnableVertexAttribArray(3);
    }

    void MeshRender::event()
    {
        bool isHovered = ImGui::IsWindowHovered();
        bool isFocused = ImGui::IsWindowFocused();
        bool isMiddleDown = ImGui::IsMouseDown(ImGuiMouseButton_Middle);
        bool isRightClick = ImGui::GetIO().MouseClicked[ImGuiMouseButton_Right];
        if (!inDrag && isHovered && isMiddleDown)
        {
            inDrag = true;
            dragX = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).x;
            dragY = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).y;
        }
        if (inDrag && !isMiddleDown)
        {
            inDrag = false;
        }
        if (inDrag)
        {
            float dragX_new = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).x;
            float dragY_new = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).y;
            auto right = camera.Right();
            camera.rotate(dragY_new - dragY, right);
            auto up = camera.Up;
            auto worldUp = glm::vec3(0.0f, 1.0f, 0.0f);
            if (glm::dot(up, worldUp) < 0)
                worldUp = -worldUp;
            camera.rotate(dragX_new - dragX, worldUp);
            dragX = dragX_new;
            dragY = dragY_new;
        }
        if (isHovered)
        {
            float wheelx = ImGui::GetIO().MouseWheel;
            camera.ProcessMouseScroll(wheelx);
        }
        auto rect_min = ImGui::GetItemRectMin();
        auto mouse_pos = ImGui::GetMousePos();
        float relative_x = (mouse_pos.x - rect_min.x) / wsize.x * 2.0f - 1.0f;
        float relative_y = (mouse_pos.y - rect_min.y) / wsize.y * 2.0f - 1.0f;
        if (isHovered && isRightClick)
        {
            auto image_pos = glm::vec4(relative_x, -relative_y, 1.0f, 1.0f);
            auto world_pos = glm::inverse(camera_projection) * image_pos;
            // LOG("image_pos: " << image_pos.x << " " << image_pos.y << " " << image_pos.z << " " << image_pos.w << std::endl);
            // LOG("world_pos: " << world_pos.x << " " << world_pos.y << " " << world_pos.z);
            world_pos.w = 1.0f;
            world_pos = glm::inverse(camera_view) * world_pos;
            // LOG("ray_target: " << world_pos.x << " " << world_pos.y << " " << world_pos.z << " " << world_pos.w << std::endl);
            GArr<float> distance(triangles_g.size());
            float3 ray_origin = make_float3(camera.Position.x, camera.Position.y, camera.Position.z);
            float3 ray_target = make_float3(world_pos.x, world_pos.y, world_pos.z);
            float3 ray_direction = ray_target - ray_origin;
            cuExecuteBlock(triangles_g.size(), CUDA_BLOCK_SIZE, ray_mesh_distance_kernel, vertices_g, triangles_g, ray_origin, ray_direction, distance);
            auto result_ptr = thrust::min_element(thrust::device, distance.begin(), distance.end());
            GArr<float> result(result_ptr, 1);
            int min_idx = result_ptr - distance.begin();
            if (result.last_item() < FLT_MAX)
            {
                if (selectedTriangle >= 0)
                {
                    meshData[selectedTriangle].flag1 = 0;
                    meshData[selectedTriangle].flag2 = 0;
                    meshData[selectedTriangle].flag3 = 0;
                    selectedTriangle = -1;
                }
                selectedTriangle = min_idx;
                meshData[selectedTriangle].flag1 = 1;
                meshData[selectedTriangle].flag2 = 1;
                meshData[selectedTriangle].flag3 = 1;
                meshNeedsUpdate = true;
                soundNeedsUpdate = true;
            }
        }
    }

    void MeshRender::load_mesh(CArr<float3> vertices_, CArr<int3> triangles_, CArr<float3> texverts_, CArr<int3> textris_)
    {
        vertices = vertices_;
        triangles = triangles_;
        // get bounding box
        float3 min_pos = vertices[0];
        float3 max_pos = vertices[0];
        for (int i = 1; i < vertices.size(); i++)
        {
            min_pos.x = min(min_pos.x, vertices[i].x);
            min_pos.y = min(min_pos.y, vertices[i].y);
            min_pos.z = min(min_pos.z, vertices[i].z);
            max_pos.x = max(max_pos.x, vertices[i].x);
            max_pos.y = max(max_pos.y, vertices[i].y);
            max_pos.z = max(max_pos.z, vertices[i].z);
        }
        // normalize vertices and move to center
        float3 center = (min_pos + max_pos) / 2.0f;
        float3 scale_f3 = (max_pos - min_pos) / 2.0f;
        float scale  = max(max(scale_f3.x, scale_f3.y), scale_f3.z);
        for (int i = 0; i < vertices.size(); i++)
        {
            vertices[i] = (vertices[i] - center) / scale;
        }
        bbox_min = make_float3(-1.0f, -1.0f, -1.0f);
        bbox_max = make_float3(1.0f, 1.0f, 1.0f);
        vertices_g.assign(vertices);
        triangles_g.assign(triangles);
        texverts_g.assign(texverts_);
        textriangles_g.assign(textris_);
        meshData_g.resize(triangles_g.size());
        GArr<float3> vert_norm;
        vert_norm.resize(vertices_g.size());
        cuExecute(triangles_g.size(), mesh_preprocess, vertices_g, triangles_g, texverts_g, textriangles_g, meshData_g, vert_norm);
        cuExecute(triangles_g.size(), mesh_preprocess_normals, triangles_g, meshData_g, vert_norm);
        meshData.assign(meshData_g);
        meshNeedsUpdate = true;
    }

    void MeshRender::resetMesh()
    {

    }

    void MeshRender::update()
    {
        ImGui::BeginChild("Render");
        resize();
        event();
        if (meshNeedsUpdate)
        {
            updateMesh();
            meshNeedsUpdate = false;
        }
        float tx = wsize.x * OVERSAMPLE, ty = wsize.y * OVERSAMPLE;
        glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
        glViewport(0, 0, tx, ty);
            glEnable(GL_DEPTH_TEST);
        glClearColor(0.3f, 0.3f, 0.3f, 0.3f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // shader.setVec3("objectColor", 0.5f, 0.5f, 0.31f);
        shader.setVec3("lightColor", 1.0f, 1.0f, 1.0f);
        shader.setVec3("selectedColor", 1.0f, 0.0f, 0.0f);

        for (int light_idx = 0; light_idx < pointLightPositions.size(); light_idx++)
        {
            shader.setVec3("lightPos[" + std::to_string(light_idx) + "]", pointLightPositions[light_idx]);
        }
        shader.setVec3("viewPos", camera.Position);
        // view/projection transformations
        float Zoom_delta = 0.0f;
        if (ty > tx)
        {
            auto fovy = glm::radians(camera.Zoom);
            auto aspect = ty / tx;
            Zoom_delta = atan(aspect * tan(fovy / 2)) * 2 - fovy;
        }
        camera_projection = glm::perspective(glm::radians(camera.Zoom) + Zoom_delta, tx / ty, 0.1f, 100.0f);
        camera_view = camera.GetViewMatrix();
        shader.setMat4("projection", camera_projection);
        shader.setMat4("view", camera_view);
        // world transformation
        glm::mat4 model = glm::mat4(1.0f);
        shader.setMat4("model", model);

        // render the mesh
        glBindVertexArray(meshVAO);
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, textureID);
        glActiveTexture(GL_TEXTURE1);
        glBindTexture(GL_TEXTURE_CUBE_MAP, skycubeID);
        glDrawArrays(GL_TRIANGLES, 0, 3 * meshData.size());
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        glDisable(GL_DEPTH_TEST);
        ImGui::Image((ImTextureID)(uintptr_t)textureColorbuffer, wsize, ImVec2(0, 1), ImVec2(1, 0));
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        ImGui::EndChild();
    }

    void MeshRender::changeMaterial(int chosenID)
    {
        const char* materialNames[] = { "Ceramic_Glazed", "Glass_Simple", "Bright_tiles_square", "Plastic_Shader", "Rough_Iron_Steel", "Stainless_Steel", "wet_aluminium"};
        if(chosenID == 1) // glass
        {
            shader.setInt("useSkyCube", 1);
            std::string cubePath = std::string(ASSET_DIR) + std::string("/materials/");
            loadSkycube(cubePath);
            return;
        }
        material = loadMaterial(std::string(ASSET_DIR) + std::string("/materials/") + mtlLib, materialNames[chosenID]);

        bool useTexture = material.texturePicName.length() != 0;
        if(useTexture)
        {
            std::string texturePath = std::string(ASSET_DIR) + std::string("/materials/") + material.texturePicName;
            loadTexture(texturePath.c_str());
        }
        std::cout <<material.ambientCoeff.x << " " << material.ambientCoeff.y<< " " << material.ambientCoeff.z<<"\n";
        shader.setVec3("ambientCoeff", material.ambientCoeff.x, material.ambientCoeff.y, material.ambientCoeff.z);
        shader.setVec3("diffuseCoeff", material.diffuseCoeff.x, material.diffuseCoeff.y, material.diffuseCoeff.z);
        shader.setVec3("specularCoeff", material.specularCoeff.x, material.specularCoeff.y, material.specularCoeff.z);
        shader.setFloat("specularExp", material.specularExp);
        shader.setFloat("alpha", material.alpha);
        shader.setInt("useTexture", (int)useTexture);
        shader.setInt("useSkyCube", 0);
        std::cout << material.alpha << "\n";
        return;
    }

} // namespace  SoundRender
