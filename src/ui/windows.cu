#include "hip/hip_runtime.h"
#include "window.h"
#include "gui_kernel.h"
#include "stb_image.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

namespace SoundRender
{
    void MeshRender::loadTexture(const char* path)
    {
        glGenTextures(1, &textureID);
        int width, height, channels;
        unsigned char* data = stbi_load(path, &width, &height, &channels, 0);
        if(data == nullptr)
        {
            std::cout << "Texture failed to load at path : " << path << std::endl;
            stbi_image_free(data);
            textureID = 0;
        }
        GLenum format;
        switch (channels)
        {
        case 1:
            format = GL_RED;
            break;
        case 3:
            format = GL_RGB;
            break;
        case 4:
            format = GL_RGBA; 
            break;
        }
        glBindTexture(GL_TEXTURE_2D, textureID);
        glTexImage2D(GL_TEXTURE_2D, 0, format, width, height, 0, format, GL_UNSIGNED_BYTE, data);
        // glGenerateMipmap(GL_TEXTURE_2D);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        stbi_image_free(data);
        glBindTexture(GL_TEXTURE_2D, 0);
        return;
    }

    void MeshRender::Prepare(std::string mtlLibName)
    {
        mtlLib = mtlLibName;
        return;
    }

    void MeshRender::init()
    {
        material = loadMaterial(std::string(ASSET_DIR) + std::string("/materials/") + mtlLib, "Ceramic_Glazed");
        bool useTexture = material.texturePicName.length() != 0;
        if(useTexture)
        {
            std::string texturePath = std::string(ASSET_DIR) + std::string("/materials/") + material.texturePicName;
            loadTexture(texturePath.c_str());
        }
        glGenFramebuffers(1, &framebuffer);
        glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
        // glEnable(GL_TEXTURE_2D);
        glGenTextures(1, &textureColorbuffer);
        glBindTexture(GL_TEXTURE_2D, textureColorbuffer);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, 1000, 1000, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, textureColorbuffer, 0);
        glBindTexture(GL_TEXTURE_2D, 0);
        
        glGenRenderbuffers(1, &rbo);
        glBindRenderbuffer(GL_RENDERBUFFER, rbo);
        glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, 1000, 1000);
        glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_STENCIL_ATTACHMENT, GL_RENDERBUFFER, rbo);
        glBindRenderbuffer(GL_RENDERBUFFER, 0);
        if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
            LOG_ERROR("Framebuffer not complete!")
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        shader.load(std::string(SHADER_DIR) + std::string("/mesh.vert"),
                    std::string(SHADER_DIR) + std::string("/mesh.frag"));
        shader.use();
        shader.setInt("Texture", 0);
        shader.setVec3("ambientCoeff", material.ambientCoeff.x, material.ambientCoeff.y, material.ambientCoeff.z);
        shader.setVec3("diffuseCoeff", material.diffuseCoeff.x, material.diffuseCoeff.y, material.diffuseCoeff.z);
        shader.setVec3("specularCoeff", material.specularCoeff.x, material.specularCoeff.y, material.specularCoeff.z);
        shader.setFloat("specularExp", material.specularExp);
        shader.setFloat("alpha", material.alpha);
        shader.setInt("useTexture", (int)useTexture);
    }

    void MeshRender::resize()
    {
        ImVec2 size = ImGui::GetWindowSize();
        if (size.x != wsize.x || size.y != wsize.y)
        {
            wsize = size;
            float tx = wsize.x * OVERSAMPLE, ty = wsize.y * OVERSAMPLE;
            glBindTexture(GL_TEXTURE_2D, textureColorbuffer);
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, tx, ty, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
            glBindTexture(GL_TEXTURE_2D, 0);
            glBindRenderbuffer(GL_RENDERBUFFER, rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH24_STENCIL8, tx, ty);
            glBindRenderbuffer(GL_RENDERBUFFER, 0);
        }
    }

    void MeshRender::updateMesh()
    {
        glGenVertexArrays(1, &meshVAO);
        glGenBuffers(1, &meshVBO);
        glBindVertexArray(meshVAO);
        glBindBuffer(GL_ARRAY_BUFFER, meshVBO);
        glBufferData(GL_ARRAY_BUFFER, sizeof(Triangle) * meshData.size(), meshData.data(), GL_STATIC_DRAW);
        // position attribute
        glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void *)0);
        glEnableVertexAttribArray(0);
        // normal attribute
        glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void *)(3 * sizeof(float)));
        glEnableVertexAttribArray(1);
        // index attribute
        glVertexAttribIPointer(2, 1, GL_INT, 9 * sizeof(float) + sizeof(int), (void *)(9 * sizeof(float)));
        glEnableVertexAttribArray(2);
        // texture attribute
        glVertexAttribPointer(3, 3, GL_FLOAT, GL_FALSE, 9 * sizeof(float) + sizeof(int), (void*)(6 * sizeof(float)));
        glEnableVertexAttribArray(3);
    }

    void MeshRender::event()
    {
        bool isHovered = ImGui::IsWindowHovered();
        bool isFocused = ImGui::IsWindowFocused();
        bool isMiddleDown = ImGui::IsMouseDown(ImGuiMouseButton_Middle);
        bool isRightClick = ImGui::GetIO().MouseClicked[ImGuiMouseButton_Right];
        if (!inDrag && isHovered && isMiddleDown)
        {
            inDrag = true;
            dragX = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).x;
            dragY = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).y;
        }
        if (inDrag && !isMiddleDown)
        {
            inDrag = false;
        }
        if (inDrag)
        {
            float dragX_new = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).x;
            float dragY_new = ImGui::GetMouseDragDelta(ImGuiMouseButton_Middle).y;
            auto right = camera.Right();
            camera.rotate(dragY_new - dragY, right);
            auto up = camera.Up;
            auto worldUp = glm::vec3(0.0f, 1.0f, 0.0f);
            if (glm::dot(up, worldUp) < 0)
                worldUp = -worldUp;
            camera.rotate(dragX_new - dragX, worldUp);
            dragX = dragX_new;
            dragY = dragY_new;
        }
        if (isHovered)
        {
            float wheelx = ImGui::GetIO().MouseWheel;
            camera.ProcessMouseScroll(wheelx);
        }
        auto rect_min = ImGui::GetItemRectMin();
        auto mouse_pos = ImGui::GetMousePos();
        float relative_x = (mouse_pos.x - rect_min.x) / wsize.x * 2.0f - 1.0f;
        float relative_y = (mouse_pos.y - rect_min.y) / wsize.y * 2.0f - 1.0f;
        if (isHovered && isRightClick)
        {
            auto image_pos = glm::vec4(relative_x, -relative_y, 1.0f, 1.0f);
            auto world_pos = glm::inverse(camera_projection) * image_pos;
            // LOG("image_pos: " << image_pos.x << " " << image_pos.y << " " << image_pos.z << " " << image_pos.w << std::endl);
            // LOG("world_pos: " << world_pos.x << " " << world_pos.y << " " << world_pos.z);
            world_pos.w = 1.0f;
            world_pos = glm::inverse(camera_view) * world_pos;
            // LOG("ray_target: " << world_pos.x << " " << world_pos.y << " " << world_pos.z << " " << world_pos.w << std::endl);
            GArr<float> distance(triangles_g.size());
            float3 ray_origin = make_float3(camera.Position.x, camera.Position.y, camera.Position.z);
            float3 ray_target = make_float3(world_pos.x, world_pos.y, world_pos.z);
            float3 ray_direction = ray_target - ray_origin;
            cuExecuteBlock(triangles_g.size(), CUDA_BLOCK_SIZE, ray_mesh_distance_kernel, vertices_g, triangles_g, ray_origin, ray_direction, distance);
            auto result_ptr = thrust::min_element(thrust::device, distance.begin(), distance.end());
            GArr<float> result(result_ptr, 1);
            int min_idx = result_ptr - distance.begin();
            if (result.last_item() < FLT_MAX)
            {
                if (selectedTriangle >= 0)
                {
                    meshData[selectedTriangle].flag1 = 0;
                    meshData[selectedTriangle].flag2 = 0;
                    meshData[selectedTriangle].flag3 = 0;
                    selectedTriangle = -1;
                }
                selectedTriangle = min_idx;
                meshData[selectedTriangle].flag1 = 1;
                meshData[selectedTriangle].flag2 = 1;
                meshData[selectedTriangle].flag3 = 1;
                meshNeedsUpdate = true;
                soundNeedsUpdate = true;
            }
        }
    }

    void MeshRender::load_mesh(CArr<float3> vertices_, CArr<int3> triangles_, CArr<float3> texverts_, CArr<int3> textris_)
    {
        vertices = vertices_;
        triangles = triangles_;
        // get bounding box
        float3 min_pos = vertices[0];
        float3 max_pos = vertices[0];
        for (int i = 1; i < vertices.size(); i++)
        {
            min_pos.x = min(min_pos.x, vertices[i].x);
            min_pos.y = min(min_pos.y, vertices[i].y);
            min_pos.z = min(min_pos.z, vertices[i].z);
            max_pos.x = max(max_pos.x, vertices[i].x);
            max_pos.y = max(max_pos.y, vertices[i].y);
            max_pos.z = max(max_pos.z, vertices[i].z);
        }
        // normalize vertices and move to center
        float3 center = (min_pos + max_pos) / 2.0f;
        float3 scale_f3 = (max_pos - min_pos) / 2.0f;
        float scale  = max(max(scale_f3.x, scale_f3.y), scale_f3.z);
        for (int i = 0; i < vertices.size(); i++)
        {
            vertices[i] = (vertices[i] - center) / scale;
        }
        bbox_min = make_float3(-1.0f, -1.0f, -1.0f);
        bbox_max = make_float3(1.0f, 1.0f, 1.0f);
        vertices_g.assign(vertices);
        triangles_g.assign(triangles);
        texverts_g.assign(texverts_);
        textriangles_g.assign(textris_);
        meshData_g.resize(triangles_g.size());
        cuExecuteBlock(triangles_g.size(), CUDA_BLOCK_SIZE, mesh_preprocess, vertices_g, triangles_g, texverts_g, textriangles_g, meshData_g);
        meshData.assign(meshData_g);
        meshNeedsUpdate = true;
    }

    void MeshRender::resetMesh()
    {
        meshData_g.resize(triangles_g.size());
        cuExecuteBlock(triangles_g.size(), CUDA_BLOCK_SIZE, mesh_preprocess, vertices_g, triangles_g, texverts_g, textriangles_g, meshData_g);
        meshData.assign(meshData_g);
        meshNeedsUpdate = true;
        if (selectedTriangle >= 0)
        {
            meshData[selectedTriangle].flag1 = 0;
            meshData[selectedTriangle].flag2 = 0;
            meshData[selectedTriangle].flag3 = 0;
            selectedTriangle = -1;
        }
    }

    void MeshRender::update()
    {
        ImGui::BeginChild("Render");
        resize();
        event();
        if (meshNeedsUpdate)
        {
            updateMesh();
            meshNeedsUpdate = false;
        }
        float tx = wsize.x * OVERSAMPLE, ty = wsize.y * OVERSAMPLE;
        glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
        glViewport(0, 0, tx, ty);
        struct AlphaTest
        {
            int id;
            float dis;
            bool operator<(const AlphaTest& t2)const {return dis < t2.dis;}
        };
        std::vector<AlphaTest> v(triangles.size());
        for(int i = 0; i < v.size(); i++)
        {
            v[i].id = i;
            auto temp = (vertices[triangles[i].x] + vertices[triangles[i].y] + vertices[triangles[i].z]) / 3;
            v[i].dis = glm::length(camera.Position - glm::vec3{temp.x, temp.y, temp.z});
        }
        std::set<AlphaTest> sortedArray(v.begin(), v.end());
        if(material.alpha < 0.99)
        {
            glEnable(GL_BLEND);
            glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
            glDisable(GL_DEPTH_TEST);
        }
        else
            glEnable(GL_DEPTH_TEST);
        glClearColor(0.3f, 0.3f, 0.3f, 0.3f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // shader.setVec3("objectColor", 0.5f, 0.5f, 0.31f);
        shader.setVec3("lightColor", 1.0f, 1.0f, 1.0f);
        shader.setVec3("selectedColor", 1.0f, 0.0f, 0.0f);

        for (int light_idx = 0; light_idx < pointLightPositions.size(); light_idx++)
        {
            shader.setVec3("lightPos[" + std::to_string(light_idx) + "]", pointLightPositions[light_idx]);
        }
        shader.setVec3("viewPos", camera.Position);
        // view/projection transformations
        float Zoom_delta = 0.0f;
        if (ty > tx)
        {
            auto fovy = glm::radians(camera.Zoom);
            auto aspect = ty / tx;
            Zoom_delta = atan(aspect * tan(fovy / 2)) * 2 - fovy;
        }
        camera_projection = glm::perspective(glm::radians(camera.Zoom) + Zoom_delta, tx / ty, 0.1f, 100.0f);
        camera_view = camera.GetViewMatrix();
        shader.setMat4("projection", camera_projection);
        shader.setMat4("view", camera_view);
        // world transformation
        glm::mat4 model = glm::mat4(1.0f);
        shader.setMat4("model", model);

        // render the mesh
        glBindVertexArray(meshVAO);
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, textureID);
        if(material.alpha < 0.99)
        {
            for(auto& it: sortedArray)
            {
                glDrawArrays(GL_TRIANGLES, it.id * 3, 3);
            }
        }
        else
            glDrawArrays(GL_TRIANGLES, 0, 3 * meshData.size());
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        glDisable(GL_DEPTH_TEST);
        ImGui::Image((ImTextureID)(uintptr_t)textureColorbuffer, wsize, ImVec2(0, 1), ImVec2(1, 0));
        glBindFramebuffer(GL_FRAMEBUFFER, 0);
        ImGui::EndChild();
    }

    void MeshRender::changeMaterial(int chosenID)
    {
        // const char* items[] = { "Ceramic", "Glass", "Wood", "Plastic", "Iron", "Polycarbonate", "Steel", "Tin"};
        const char* materialNames[] = { "Ceramic_Glazed", "Glass_Simple", "Bright_tiles_square", "Plastic_Shader", "Rough_Iron_Steel", "Polycarbonate", "Stainless_Steel", "wet_aluminium"};
        material = loadMaterial(std::string(ASSET_DIR) + std::string("/materials/") + mtlLib, materialNames[chosenID]);
        bool useTexture = material.texturePicName.length() != 0;
        if(useTexture)
        {
            std::string texturePath = std::string(ASSET_DIR) + std::string("/materials/") + material.texturePicName;
            loadTexture(texturePath.c_str());
        }
        std::cout <<material.ambientCoeff.x << " " << material.ambientCoeff.y<< " " << material.ambientCoeff.z<<"\n";
        shader.setVec3("ambientCoeff", material.ambientCoeff.x, material.ambientCoeff.y, material.ambientCoeff.z);
        shader.setVec3("diffuseCoeff", material.diffuseCoeff.x, material.diffuseCoeff.y, material.diffuseCoeff.z);
        shader.setVec3("specularCoeff", material.specularCoeff.x, material.specularCoeff.y, material.specularCoeff.z);
        shader.setFloat("specularExp", material.specularExp);
        shader.setFloat("alpha", material.alpha);
        shader.setInt("useTexture", (int)useTexture);
        std::cout << material.alpha << "\n";
        return;
    }

} // namespace  SoundRender
