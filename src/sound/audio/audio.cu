#include "hip/hip_runtime.h"
#include "audio.h"

namespace SoundRender
{

    static int patestCallback(const void *inputBuffer, void *outputBuffer,
                              unsigned long framesPerBuffer,
                              const PaStreamCallbackTimeInfo *timeInfo,
                              PaStreamCallbackFlags statusFlags,
                              void *userData)
    {
        paSoundData *data = (paSoundData *)userData;
        float *out = (float *)outputBuffer;
        for (int i = 0; i < framesPerBuffer; i++)
        {
            auto left_phase = data->left_phase % TABLE_SIZE;
            auto right_phase = data->right_phase % TABLE_SIZE;
            *out++ = data->signal[left_phase];
            *out++ = data->signal[right_phase]; /* right */
            data->left_phase++;
            data->right_phase++;
            data->signal[left_phase] = 0;
        }

        return paContinue;
    }
    AudioWapper::AudioWapper()
    {
        for (int i = 0; i < TABLE_SIZE; i++)
        {
            data.signal[i] = 0;
        }
        data.left_phase = data.right_phase = data.update_phase = 0;
        err = Pa_Initialize();
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }

        outputParameters.device = Pa_GetDefaultOutputDevice(); /* default output device */
        outputParameters.channelCount = 2;                     /* stereo output */
        outputParameters.sampleFormat = paFloat32;             /* 32 bit floating point output */
        outputParameters.suggestedLatency = Pa_GetDeviceInfo(outputParameters.device)->defaultLowOutputLatency;
        outputParameters.hostApiSpecificStreamInfo = NULL;

        err = Pa_OpenStream(
            &stream,
            NULL, /* no input */
            &outputParameters,
            SAMPLE_RATE,
            FRAMES_PER_BUFFER,
            paClipOff, /* we won't output out of range samples so don't bother clipping them */
            patestCallback,
            &data);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }

        err = Pa_StartStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
    }

    void AudioWapper::update()
    {
        float left_phase = data.left_phase;
        // ImGui::Text("Here is AudioWapper Module");
        // ImGui::Text("delta_phase: %d", (int)(data.update_phase - left_phase));
        // ImGui::Text("TABLE_SIZE: %d", TABLE_SIZE);

        // Here we store our selection data as an index.

        int sample_num;
        if (last_phase == -1)
        {
            auto fps = ImGui::GetIO().Framerate;
            data.update_phase = left_phase + DELTA_SAMPLE_NUM * 1.5;
            sample_num = (int)(1 / fps * SAMPLE_RATE);
        }
        else
        {
            sample_num = left_phase - last_phase;
        }
        last_phase = left_phase;

        float scale_factor = Correction::soundScale;
        int mode_idx = -1;
        for (auto &modalInfo : modalSound->modalInfos)
        {
            mode_idx++;
            float ffat_factor = modalSound->GetFFATFactor(modalInfo) * 10000;
            if (ffat_last_factor[mode_idx] < 0)
            {
                ffat_last_factor[mode_idx] = ffat_factor;
            }
            
            float q1 = modalInfo.q1;
            float q2 = modalInfo.q2;
            float f = modalInfo.f;
            float c1 = modalInfo.coeff1;
            float c2 = modalInfo.coeff2;
            float c3 = modalInfo.coeff3;
            
            if (abs(q1 * ffat_factor * scale_factor) < 1e-4 && abs(q2 * ffat_factor * scale_factor) < 1e-4 && f < 1e-4)
            {
                modalInfo.q1 = 0;
                modalInfo.q2 = 0;
                modalInfo.f = 0;
                ffat_last_factor[mode_idx] = ffat_factor;
                continue;
            }

            for (int i = 0; i < sample_num; i++)
            {
                float k = (float)i / (float)sample_num;
                float ffat_factor_smooth = (1 - k) * ffat_last_factor[mode_idx] + k * ffat_factor;
                float q = c1 * q1 + c2 * q2 + c3 * f;
                q2 = q1;
                q1 = q;
                f = f * 0.1;
                data.signal[(data.update_phase + i) % TABLE_SIZE] += q * ffat_factor_smooth * scale_factor;
            }
            modalInfo.q1 = q1;
            modalInfo.q2 = q2;
            modalInfo.f = f;
            ffat_last_factor[mode_idx] = ffat_factor;
        }
        // if (modalSound->click_current_frame)
        // for (int i = 0; i < signalPlotData.size; i++)
        // {
        //     signalPlotData.y[i] = data.signal[(data.update_phase + i) % TABLE_SIZE];
        //     if (i > sample_num)
        //         break;
        // }
        modalSound->click_current_frame = false;

        data.update_phase = data.update_phase + sample_num;

        // plot the sound wave
        // if (ImPlot::BeginPlot("Audio Click Signal"))
        // {
        //     ImPlot::SetupAxisLimits(ImAxis_X1, 0, 300);
        //     ImPlot::SetupAxisLimits(ImAxis_Y1, -1, 1);
        //     ImPlot::PlotLine("signal", signalPlotData.x, signalPlotData.y, sample_num);
        //     ImPlot::EndPlot();
        // }

        // ImGui::Text("sample_num: %d", sample_num);
        // end of plotting sound wave

        // plot the FFAT map
        // static ImPlotColormap map = ImPlotColormap_Viridis;
        // ImPlot::PushColormap(map);

        // const int plotRowNum = (int)modalSound->modalInfos[0].ffat.size();
        // const int plotColNum = (int)modalSound->modalInfos[0].ffat[0].size();
        // static double values[RowNum][ColNum];

        // static int modal_index = 0;
        // ImGui::SliderInt("modal index", &modal_index, 0, 19);
        // const int plotRowNum = 64, plotColNum = 32;
        // static double values[plotRowNum][plotColNum];

        // get current theta & phi , which correspond with current row & col in ffat map
        //  auto campos = modalSound->mesh_render->camera.Position * Correction::camScale;
        //  const float camx = campos[0], camy = campos[1], camz = campos[2];

        // const float r = glm::length(campos) + 1e-4f; // to prevent singular point.
        // const size_t ffatRowNum = modalSound->modalInfos[0].ffat.size();
        // const size_t ffatColNum = modalSound->modalInfos[0].ffat[0].size();
        // const float rowSampleIntervalRep = ffatRowNum / (2 * PI);
        // const float colSampleIntervalRep = ffatColNum / PI;

        // float theta = std::acos(camz / r);
        // float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= -1e-5f && camy >= -1e-5f ? 0.0f : std::fmod(std::atan2(camy, camx) + 2 * PI, 2 * PI);

        // float colInter = theta * colSampleIntervalRep, rowInter = phi * rowSampleIntervalRep;
        // int col = static_cast<int>(colInter);
        // int row = static_cast<int>(rowInter);

        // int ffat_i=0,ffat_j=0;
        // for (int i = 0; i < plotRowNum; i++)
        // {
        //     for (int j = 0; j < plotColNum; j++)
        //     {
        //         //always put the coordinate (theta,phi) in the center of the FFAT map
        //         ffat_i=(i+plotRowNum/2-row+plotRowNum)%plotRowNum;
        //         ffat_j=(j+plotColNum/2-col+plotColNum)%plotColNum;
        //         values[i][j] = modalSound->modalInfos[modal_index].ffat[ffat_i][ffat_j];
        //     }
        // }

        // static float scale_min = 0;
        // static float scale_max = 0.001f;
        // ImGui::SetNextItemWidth(225);
        // ImGui::DragFloatRange2("Min / Max", &scale_min, &scale_max, 0.001f, 0, 1);

        // if (ImPlot::BeginPlot("##Heatmap",ImVec2(225,450)))
        // {

        //     ImPlot::SetupAxes(NULL, NULL, ImPlotAxisFlags_NoDecorations, ImPlotAxisFlags_NoDecorations);
        //     ImPlot::PlotHeatmap("ffat map", values[0],plotRowNum, plotColNum, scale_min, scale_max, NULL);
        //     ImPlot::EndPlot();
        // }
        // ImGui::SameLine();
        // ImPlot::ColormapScale("##HeatScale", scale_min, scale_max, ImVec2(60, 225));
        // ImPlot::PopColormap();
        // end of plotting FFAT map
    }

    void AudioWapper::close()
    {
        auto err = Pa_StopStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
        err = Pa_CloseStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
        err = Pa_Terminate();
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
        }
    }
}