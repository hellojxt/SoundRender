#include "hip/hip_runtime.h"
#include "audio.h"


namespace SoundRender
{
    

    static int patestCallback(const void *inputBuffer, void *outputBuffer,
                              unsigned long framesPerBuffer,
                              const PaStreamCallbackTimeInfo *timeInfo,
                              PaStreamCallbackFlags statusFlags,
                              void *userData)
    {
        paSoundData *data = (paSoundData *)userData;
        float *out = (float *)outputBuffer;
        for (int i = 0; i < framesPerBuffer; i++)
        {
            auto left_phase = data->left_phase % TABLE_SIZE;
            auto right_phase = data->right_phase % TABLE_SIZE;
            *out++ = data->signal[left_phase];
            *out++ = data->signal[right_phase]; /* right */
            data->left_phase++;
            data->right_phase++;
            data->signal[left_phase] = 0;
        }

        return paContinue;
    }
    AudioWapper::AudioWapper()
    {
        for (int i = 0; i < TABLE_SIZE; i++)
        {
            data.signal[i] = 0;
        }
        data.left_phase = data.right_phase = data.update_phase = 0;
        err = Pa_Initialize();
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }

        outputParameters.device = Pa_GetDefaultOutputDevice(); /* default output device */
        outputParameters.channelCount = 2;                     /* stereo output */
        outputParameters.sampleFormat = paFloat32;             /* 32 bit floating point output */
        outputParameters.suggestedLatency = Pa_GetDeviceInfo(outputParameters.device)->defaultLowOutputLatency;
        outputParameters.hostApiSpecificStreamInfo = NULL;

        err = Pa_OpenStream(
            &stream,
            NULL, /* no input */
            &outputParameters,
            SAMPLE_RATE,
            FRAMES_PER_BUFFER,
            paClipOff, /* we won't output out of range samples so don't bother clipping them */
            patestCallback,
            &data);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }

        err = Pa_StartStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
    }

    void AudioWapper::update()
    {
        ImGui::Text("Here is AudioWapper Module");

        float left_phase = data.left_phase;
        ImGui::Text("delta_phase: %d", (int)(data.update_phase - left_phase));
        ImGui::Text("TABLE_SIZE: %d", TABLE_SIZE);


        static int item_current_idx = 0; // Here we store our selection data as an index.
        if (ImGui::BeginListBox("Material List:\n"))
        {
            for (int n = 0; n < IM_ARRAYSIZE(MaterialConst::names); n++)
            {
                const bool is_selected = (item_current_idx == n);
                if (ImGui::Selectable(MaterialConst::names[n].c_str(), is_selected))
                {
                    if (item_current_idx != n)
                    {
                        item_current_idx = n;
                        modalSound->init(modalSound->filename, item_current_idx);
                        modalSound->SetMaterial(item_current_idx, true);
                    }
                    
                }
                // Set the initial focus when opening the combo (scrolling + keyboard navigation focus)
                if (is_selected)
                    ImGui::SetItemDefaultFocus();
            }
            ImGui::EndListBox();
        }


        int sample_num;
        if (last_phase == -1)
        {
            auto fps = ImGui::GetIO().Framerate;
            data.update_phase = left_phase + DELTA_SAMPLE_NUM *1.5;
            sample_num = (int)(1 / fps * SAMPLE_RATE);
        }
        else
        {
            sample_num = left_phase - last_phase;
        }
        last_phase = left_phase;

        if (modalSound->click_current_frame)
        {
            float scale_factor = Correction::soundScale;
            for (auto &modalInfo : modalSound->modalInfos)
            {
                float ffat_factor = modalSound->GetFFATFactor(modalInfo) * 10000;
                float q1 = modalInfo.q1;
                float q2 = modalInfo.q2;
                float f = modalInfo.f;
                float c1 = modalInfo.coeff1;
                float c2 = modalInfo.coeff2;
                float c3 = modalInfo.coeff3;

                if (abs(q1 * ffat_factor * scale_factor) < 1e-3 && abs(q2 * ffat_factor * scale_factor) < 1e-3 && f < 1e-3)
                {
                    continue;
                }

                for (int i = 0; i < sample_num; i++)
                {
                    float q = c1 * q1 + c2 * q2 + c3 * f;
                    q2 = q1;
                    q1 = q;
                    f = f * 0.1;
                    data.signal[(data.update_phase + i) % TABLE_SIZE] += q * ffat_factor * scale_factor;
                }
                modalInfo.q1 = q1;
                modalInfo.q2 = q2;
                modalInfo.f = f;
            }
        
            for (int i = 0; i < signalPlotData.size; i++)
            {
                signalPlotData.y[i] = data.signal[(data.update_phase + i - 100) % TABLE_SIZE];
            }
            modalSound->click_current_frame = false;
        }
        data.update_phase = data.update_phase + sample_num;
        

        // plot the sound wave
        if (ImPlot::BeginPlot("Audio Click Signal"))
        {
            ImPlot::PlotLine("signal", signalPlotData.x, signalPlotData.y, signalPlotData.size);
            ImPlot::EndPlot();
        }
        //end of plotting sound wave

        // plot the FFAT map
        static ImPlotColormap map = ImPlotColormap_Viridis;
        ImPlot::PushColormap(map);
        static int modal_index = 0;
        ImGui::SliderInt("modal index", &modal_index, 0, 19);
 
        // const int plotRowNum = (int)modalSound->modalInfos[0].ffat.size();
        // const int plotColNum = (int)modalSound->modalInfos[0].ffat[0].size();
        // static double values[RowNum][ColNum];
        const int plotRowNum = 64, plotColNum = 32;
        static double values[plotRowNum][plotColNum];

        //get current theta & phi , which correspond with current row & col in ffat map 
        auto campos = modalSound->mesh_render->camera.Position * Correction::camScale;
        const float camx = campos[0], camy = campos[1], camz = campos[2];

        const float r = glm::length(campos) + 1e-4f; // to prevent singular point.
        const size_t ffatRowNum = modalSound->modalInfos[0].ffat.size();
        const size_t ffatColNum = modalSound->modalInfos[0].ffat[0].size();
        const float rowSampleIntervalRep = ffatRowNum / (2 * PI);
        const float colSampleIntervalRep = ffatColNum / PI;

        float theta = std::acos(camz / r);
        float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= -1e-5f && camy >= -1e-5f ? 0.0f : std::fmod(std::atan2(camy, camx) + 2 * PI, 2 * PI);

        float colInter = theta * colSampleIntervalRep, rowInter = phi * rowSampleIntervalRep;
        int col = static_cast<int>(colInter);
        int row = static_cast<int>(rowInter);

        int ffat_i=0,ffat_j=0;
        for (int i = 0; i < plotRowNum; i++)
        {
            for (int j = 0; j < plotColNum; j++)
            {
                //always put the coordinate (theta,phi) in the center of the FFAT map
                ffat_i=(i+plotRowNum/2-row+plotRowNum)%plotRowNum;
                ffat_j=(j+plotColNum/2-col+plotColNum)%plotColNum;
                values[i][j] = modalSound->modalInfos[modal_index].ffat[ffat_i][ffat_j];
            }
        }

        static float scale_min = 0;
        static float scale_max = 0.001f;
        ImGui::SetNextItemWidth(225);
        ImGui::DragFloatRange2("Min / Max", &scale_min, &scale_max, 0.001f, 0, 1);

        if (ImPlot::BeginPlot("##Heatmap",ImVec2(225,450)))
        {
          
            ImPlot::SetupAxes(NULL, NULL, ImPlotAxisFlags_NoDecorations, ImPlotAxisFlags_NoDecorations);
            ImPlot::PlotHeatmap("ffat map", values[0],plotRowNum, plotColNum, scale_min, scale_max, NULL);
            ImPlot::EndPlot();
        }
        ImGui::SameLine();
        ImPlot::ColormapScale("##HeatScale", scale_min, scale_max, ImVec2(60, 225));
        ImPlot::PopColormap();
        // end of plotting FFAT map


    }

    void AudioWapper::close()
    {
        auto err = Pa_StopStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
        err = Pa_CloseStream(stream);
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
            return;
        }
        err = Pa_Terminate();
        if (err != paNoError)
        {
            printf("PortAudio error: %s\n", Pa_GetErrorText(err));
        }
    }
}