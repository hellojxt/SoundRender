#include "hip/hip_runtime.h"
#include "audio.h"


namespace SoundRender
{
    float AudioWapper::CallbackForSound(AudioWapper* audio)
    {
        return audio->_CallbackForSound();
    }

    float AudioWapper::_CallbackForSound()
    {
        auto& modalInfos = modalSound->modalInfos;

        float soundResult = 0.0f;
        for (auto &modalInfo : modalInfos)
        {
            float dotResult = 0.f;
            for (auto &restForce : restForces)
            {
                size_t id = restForce.first;
                dotResult += (modalInfo.eigenVec[id] + modalInfo.eigenVec[id + 1] +
                              modalInfo.eigenVec[id + 2]) *
                             restForce.second;
            }
            modalInfo.f = dotResult;
            const auto [p, q] = modalSound->GetModalResult(modalInfo);
            soundResult += p * q;
        }

        std::vector<size_t> willDelete;
        for (auto &restForce : restForces)
        {
            restForce.second = GetDeclinedForce(restForce.second);
            if (restForce.second < 1e-5f) // force become 0
            {
                willDelete.push_back(restForce.first);
            }
        }

        if (!willDelete.empty())
        { // clear out 0 force.
            std::lock_guard<std::mutex> _(restForceMutex);
            for (size_t item : willDelete)
            {
                restForces.erase(item);
            }
        }
        return soundResult;
    }

    inline float Lerp(float x1, float x2, float coeff)
    {
        return x1 * coeff + x2 * (1 - coeff);
    }

     std::pair<float, float> AudioWapper::GetModalResult(ModalInfo &modalInfo)
    {
        const float camx = modalSound->mesh_render->camera.Position[0], 
            camy = modalSound->mesh_render->camera.Position[1],
            camz = modalSound->mesh_render->camera.Position[2];
        const size_t ffatColNum = modalInfo.ffat[0].size();
        // Here we need row and col sample intervals are the same, otherwise changes are needed.
        const float sampleIntervalRep = ffatColNum / PI;

        float theta = std::acos(camz);
        float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= 1e-5f && camy >= 1e-5f ? 0.0f : std::atan2(camy, camx) + PI;

        float colInter = theta * sampleIntervalRep, rowInter = phi * sampleIntervalRep;
        size_t col = static_cast<size_t>(colInter);
        float colFrac = colInter - static_cast<float>(col);
        size_t row = static_cast<size_t>(rowInter);
        float rowFrac = rowInter - static_cast<float>(row);
        // bi-Lerp.
        float interResult = Lerp(Lerp(modalInfo.ffat[row][col], modalInfo.ffat[row + 1][col], rowFrac),
                                 Lerp(modalInfo.ffat[row][col + 1], modalInfo.ffat[row + 1][col + 1], rowFrac), colFrac);

        float p = interResult / (camx * camx + camy * camy + camz * camz);
        float q = modalInfo.coeff1 * modalInfo.q1 + modalInfo.coeff2 * modalInfo.q2 + modalInfo.coeff3 * modalInfo.f;
        modalInfo.q2 = modalInfo.q1, modalInfo.q1 = q; // update q.

        //return {p, q};
        return std::pair(p,q);
    };
}