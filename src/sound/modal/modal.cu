#include "hip/hip_runtime.h"
#include "modal.h"
#include "objIO.h"
#include <algorithm>

namespace SoundRender
{
    namespace Correction
    {
        float camScale = 0.0f;
        float soundScale = 0.0f;

        std::unordered_map<std::string, float> allSoundScales;
    }

    float3 GetTriangleCenter(int3 tri, CArr<float3> &vertArr)
    {
        return (vertArr[tri.x] + vertArr[tri.y] + vertArr[tri.z]) / 3.0f;
    }

    float3 GetTriangleNormal(int3 tri, CArr<float3> &vertArr)
    {
        float3 e1 = vertArr[tri.x] - vertArr[tri.y], e2 = vertArr[tri.y] - vertArr[tri.z];
        return normalize(cross(e1, e2));
    }

    ModalInfo::ModalInfo(float lambda, size_t index, cnpy::NpyArray &eigenVecs, cnpy::NpyArray &ffats)
    {
        eigenVal = lambda;
        SetMaterial(0);
        size_t rank = eigenVecs.shape[0], colNum = eigenVecs.shape[1];
        eigenVec.reserve(rank);
        float *eigenVecsData = eigenVecs.data<float>();
        for (size_t i = 0; i < rank; i++)
        {
            eigenVec.push_back(eigenVecsData[i * colNum + index]);
        }

        size_t ffatRowNum = ffats.shape[1], ffatColNum = ffats.shape[2];
        double *ffatsData = ffats.data<double>() + ffatRowNum * ffatColNum * index;
        ffat.reserve(ffatRowNum);
        for (size_t i = 0; i < ffatRowNum; i++)
        {
            size_t baseIndex = i * ffatColNum;
            ffat.emplace_back(ffatsData + baseIndex, ffatsData + baseIndex + ffatColNum);
        }
        f = q1 = q2 = 0;
    }

    void ModalInfo::SetMaterial(int chosenID)
    {
        float alpha = MaterialConst::alpha[chosenID], beta = MaterialConst::beta[chosenID];
        float lambda = eigenVal;

        float omega = std::sqrt(lambda);
        float ksi = (alpha + beta * lambda) / (2 * omega);
        float omega_prime = omega * std::sqrt(1 - ksi * ksi);
        float epsilon = std::exp(-ksi * omega * MaterialConst::timestep);
        float sqrEpsilon = epsilon * epsilon;
        float theta = omega_prime * MaterialConst::timestep;
        float gamma = std::asin(ksi);

        coeff1 = 2 * epsilon * std::cos(theta);
        coeff2 = -sqrEpsilon;

        float coeff3_item1 = epsilon * std::cos(theta + gamma);
        float coeff3_item2 = sqrEpsilon * std::cos(2 * theta + gamma);
        coeff3 = 2 * (coeff3_item1 - coeff3_item2) / (3 * omega * omega_prime);
        return;
    };

    void ModalSound::init(const std::string &filename, int materialID)
    {
        this->filename = filename;
        auto pos1 = filename.rfind('/') + 1, pos2 = filename.rfind('.');
        auto modelName = filename.substr(pos1, pos2 - pos1);
        auto eigenPath = std::string(ASSET_DIR) + std::string("/eigen/") + modelName + std::string("_") + MaterialConst::names[materialID] + std::string(".npz");
        auto ffatPath = std::string(ASSET_DIR) + std::string("/acousticMap/") + modelName + std::string("_") + MaterialConst::names[materialID] + std::string(".npz");
        auto voxelPath = std::string(ASSET_DIR) + std::string("/voxel/") + modelName + std::string(".npy");
        SetModal(eigenPath.c_str(), ffatPath.c_str(), voxelPath.c_str());
        Correction::soundScale = Correction::allSoundScales[modelName];
        return;
    }

    void ModalSound::update()
    {
        ImGui::Text("Here is ModalSound Module");
        ImGui::Text("Mesh has %d vertices and %d triangles", mesh_render->vertices.size(), mesh_render->triangles.size());
        ImGui::Text("Camera position:  (%f, %f, %f)", mesh_render->camera.Position.x * Correction::camScale,
                    mesh_render->camera.Position.y * Correction::camScale, mesh_render->camera.Position.z * Correction::camScale);
        ImGui::SliderFloat("Click Force", &force, 0.0f, 1.0f);
        ImGui::Text("Force: %f", force);
        ImGui::Text("Selected Triangle Index: %d", mesh_render->selectedTriangle);
        ImGui::Text("This material & model preprocessed for %lf seconds.", preprocessTime);
        float tanHalfFov = std::tan(glm::radians(mesh_render->camera.Zoom) / 2);
        static float initTanHalfFov = tanHalfFov;
        Correction::camScale = tanHalfFov / initTanHalfFov;

        // if click or space key is pressed
        if ((mesh_render->soundNeedsUpdate || ImGui::IsKeyPressed(GLFW_KEY_SPACE)) && mesh_render->selectedTriangle != -1)
        {
            std::cout << "True2.\n";
            int3 tri = mesh_render->triangles[mesh_render->selectedTriangle];
            select_point = GetTriangleCenter(tri, mesh_render->vertices);
            auto norm = GetTriangleNormal(tri, mesh_render->vertices);
            auto y = norm.x;
            auto x = -norm.y;
            norm.x = x;
            norm.y = y;
            select_voxel_idx = GetNormalizedID(select_point);

            for (int i = 0; i < 8; i++)
            {
                auto offset = MaterialConst::offsets[i];
                auto id = vertData[select_voxel_idx.x + offset[0]][select_voxel_idx.y + offset[1]][select_voxel_idx.z + offset[2]] * 3;
                select_voxel_vertex_idx[i] = id;
                for (auto &modalInfo : modalInfos)
                {
                    auto mode_f = modalInfo.eigenVec[id] * norm.x + modalInfo.eigenVec[id + 1] * norm.y + modalInfo.eigenVec[id + 2] * norm.z;
                    modalInfo.f += mode_f * force / 8;
                }
            }
            click_current_frame = true;
            mesh_render->soundNeedsUpdate = false;
        }
        ImGui::Text("Selected Triangle Center: (%f, %f, %f)", select_point.x, select_point.y, select_point.z);
        ImGui::Text("Selected Voxel Index: (%d, %d, %d)", select_voxel_idx.x, select_voxel_idx.y, select_voxel_idx.z);
        if (select_voxel_idx.x >= 0 && select_voxel_idx.y >= 0 && select_voxel_idx.z >= 0)
            ImGui::Text("Selected Voxel value: %d", voxelData(select_voxel_idx.x, select_voxel_idx.y, select_voxel_idx.z));
        ImGui::Text("Selected Voxel Vertex Index: ");
        for (int i = 0; i < 8; i++)
        {
            ImGui::Text("%d", select_voxel_vertex_idx[i]);
        }
    }

    void ModalSound::AdjustSoundScale()
    {
        float currMax = 0.0f;
        float scale_factor = (2 * M_PI * 3000) * (2 * M_PI * 3000);
        float tanHalfFov = std::tan(glm::radians(15.0f) / 2);
        float tempScale = tanHalfFov / std::tan(glm::radians(45.0f) / 2);
        std::vector<double> ffatFactors(modalInfos.size());
        for (int i = 0; i < ffatFactors.size(); i++)
        {
            double tempMax = 0.0f;
            for (auto &row : modalInfos[i].ffat)
            {
                auto temp = std::max_element(row.begin(), row.end());
                tempMax = std::max(*temp, tempMax);
            }
            ffatFactors[i] = tempMax / (4 * tempScale);
        }
            for (int i = 0; i < mesh_render->triangles.size(); i++)
            {
                int3 tri = mesh_render->triangles[i];
                select_point = GetTriangleCenter(tri, mesh_render->vertices);
                auto norm = GetTriangleNormal(tri, mesh_render->vertices);
                select_voxel_idx = GetNormalizedID(select_point);

                for (int i = 0; i < 8; i++)
                {
                    auto offset = MaterialConst::offsets[i];
                    auto id = vertData[select_voxel_idx.x + offset[0]][select_voxel_idx.y + offset[1]][select_voxel_idx.z + offset[2]] * 3;
                    select_voxel_vertex_idx[i] = id;
                    for (auto &modalInfo : modalInfos)
                    {
                        auto mode_f = modalInfo.eigenVec[id] * norm.x + modalInfo.eigenVec[id + 1] * norm.y + modalInfo.eigenVec[id + 2] * norm.z;
                        modalInfo.f += mode_f * 1.0f / 8; // force_max = 1.0f;
                    }
                }

                float result = 0.0f;
                for (int i = 0; i < modalInfos.size(); i++)
                {
                    auto &modalInfo = modalInfos[i];
                    float ffat_factor = ffatFactors[i] * 10000;
                    float q1 = modalInfo.q1;
                    float q2 = modalInfo.q2;
                    float f = modalInfo.f;
                    float c1 = modalInfo.coeff1;
                    float c2 = modalInfo.coeff2;
                    float c3 = modalInfo.coeff3;
                    float q = c1 * q1 + c2 * q2 + c3 * f;
                    modalInfo.f = 0;
                    result += q * ffat_factor * scale_factor;
                }

                if (result > currMax)
                    currMax = result;
            }
        Correction::soundScale = scale_factor / (currMax + 0.1f);
    }

    int3 ModalSound::GetNormalizedID(float3 center)
    {
        auto y = center.x;
        auto x = -center.y;
        center.x = x;
        center.y = y;
        size_t voxelNum = voxelData.batchs - 1;
        float3 bbMin = mesh_render->bbox_min, bbMax = mesh_render->bbox_max;
        float3 relative_coord = (center - bbMin) / (bbMax - bbMin);
        return make_int3((relative_coord * (float)voxelNum));
    }

    void ModalSound::FillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        float *valueData = rawEigenValues.data<float>();
        for (int i = 0; i < selectNum; i++)
        {
            modalInfos.emplace_back(valueData[i], i, rawEigenVecs, rawFFAT);
        }
        return;
    }

    void ModalSound::FillVertID(cnpy::NpyArray &rawVoxelData)
    {
        int *voxelDataPointer = rawVoxelData.data<int>();
        size_t xSize = rawVoxelData.shape[0], ySize = rawVoxelData.shape[1],
               zSize = rawVoxelData.shape[2];

        // allocate memory.
        voxelData = CArr3D<int>(xSize, ySize, zSize, voxelDataPointer);
        vertData.resize(xSize + 1, ySize + 1, zSize + 1);
        vertData.reset();

        // check voxel in object.
        for (size_t i = 0; i < xSize; i++)
        {
            for (size_t j = 0; j < ySize; j++)
            {
                for (size_t k = 0; k < zSize; k++)
                {
                    if (voxelData(i, j, k) == 1)
                    {
                        for (auto &offset : MaterialConst::offsets)
                        {
                            vertData(i + offset[0], j + offset[1], k + offset[2]) = 1;
                        }
                    }
                }
            }
        }

        // put id in vert bucket.
        int cnt = 0;
        for (size_t i = 0; i < xSize + 1; i++)
        {
            for (size_t j = 0; j < ySize + 1; j++)
            {
                for (size_t k = 0; k < zSize + 1; k++)
                {
                    if (vertData(i, j, k) == 1)
                    {
                        vertData(i, j, k) = cnt;
                        cnt++;
                    }
                }
            }
        }
        return;
    }

    void ModalSound::SetMaterial(int chosenID, bool needShade)
    {
        LOG("SetMaterial: " << chosenID);
        for (auto &modalInfo : modalInfos)
            modalInfo.SetMaterial(chosenID);
        if(needShade)
            mesh_render->changeMaterial(chosenID);
        return;
    }

    void ModalSound::SetModal(const char *eigenPath, const char *ffatPath, const char *voxelPath)
    {
        click_current_frame = false;
        cnpy::npz_t eigenData = cnpy::npz_load(eigenPath);
        cnpy::NpyArray &rawEigenValues = eigenData["vals"];             // get S
        cnpy::NpyArray &rawEigenVecs = eigenData["vecs"];               // get U
        cnpy::NpyArray rawFFAT = cnpy::npz_load(ffatPath, "feats_out"); // get FFAT.
        assert(rawFFAT.word_size == sizeof(double));
        assert(rawEigenValues.word_size == sizeof(float));
        assert(rawEigenVecs.word_size == sizeof(float));

        FillModalInfos(rawEigenValues, rawEigenVecs, rawFFAT);
        cnpy::NpyArray rawVoxelData = cnpy::npy_load(voxelPath);
        assert(rawVoxelData.word_size == sizeof(int));
        FillVertID(rawVoxelData);
        select_voxel_idx = make_int3(-1, -1, -1);
        select_point = make_float3(-1, -1, -1);
        force = 0.5;
        for (int i = 0; i < 8; i++)
        {
            select_voxel_vertex_idx[i] = -1;
        }
        cnpy::NpyArray preTime = cnpy::npz_load(ffatPath, "time");
        assert(preTime.word_size == sizeof(double));
        preprocessTime = preTime.data<double>()[0];
        return;
    }

    inline float Lerp(float x1, float x2, float coeff)
    {
        return x1 * coeff + x2 * (1 - coeff);
    }

    float ModalSound::GetFFATFactor(ModalInfo &modalInfo)
    {
        auto campos = mesh_render->camera.Position * Correction::camScale;
        const float camx = campos[0], camy = campos[1], camz = campos[2];

        const float r = glm::length(campos) + 1e-4f; // to prevent singular point.
        const size_t ffatRowNum = modalInfo.ffat.size();
        const size_t ffatColNum = modalInfo.ffat[0].size();
        const float rowSampleIntervalRep = ffatRowNum / (2 * PI);
        const float colSampleIntervalRep = ffatColNum / PI;

        float theta = std::acos(camz / r);
        float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= -1e-5f && camy >= -1e-5f ? 0.0f : std::fmod(std::atan2(camy, camx) + 2 * PI, 2 * PI);

        float colInter = theta * colSampleIntervalRep, rowInter = phi * rowSampleIntervalRep;
        int col = static_cast<int>(colInter);
        float colFrac = colInter - static_cast<float>(col);
        if (col < 0 || col > ffatColNum)
        {
            col = 0;
            colFrac = 0;
        }
        int row = static_cast<int>(rowInter);
        float rowFrac = rowInter - static_cast<float>(row);
        if (row < 0 || row > ffatRowNum)
        {
            row = 0;
            rowFrac = 0;
        }
        // std::cout << row << " " << row + 1 << " " << col << " " << col + 1 << "\n";
        // bi-Lerp.
        int nextRow = (row + 1) % ffatRowNum, nextCol = (col + 1) % ffatColNum;
        float interResult = Lerp(Lerp(modalInfo.ffat[row][col], modalInfo.ffat[nextRow][col], rowFrac),
                                 Lerp(modalInfo.ffat[row][nextCol], modalInfo.ffat[nextRow][nextCol], rowFrac), colFrac);

        return interResult / r;
    }

    void ModalSound::PreprocessAllModals(const std::filesystem::path &meshRootPath, const std::filesystem::path &scaleFilePath)
    {
        int cnt = std::distance(std::filesystem::directory_iterator(meshRootPath), std::filesystem::directory_iterator());
        int preprocessedCnt = 0;
        std::cout << "Totally " << cnt << " files in /meshes.\n";

        std::fstream correctionFile(scaleFilePath, std::ios::in | std::ios::out | std::ios::app);
        std::string modelName;
        float modalScale;
        while (correctionFile >> modelName >> modalScale)
        {
            Correction::allSoundScales.emplace(modelName, modalScale);
        };
        correctionFile.clear();

        [[maybe_unused]] const auto AsciiWStrToStr = [](const std::wstring &wstr)
        { return std::string(wstr.begin(), wstr.end()); };
        for (const auto &entry : std::filesystem::directory_iterator(meshRootPath))
        {
#ifdef _WIN32
            auto tempName = entry.path().filename().replace_extension(L"");
            modelName = AsciiWStrToStr(tempName.c_str());
#else
            modelName = entry.path().filename().replace_extension("");
#endif
            if (Correction::allSoundScales.find(modelName) == Correction::allSoundScales.end())
            {
#ifdef _WIN32
                auto meshPath = AsciiWStrToStr(entry.path().c_str());
#else
                auto meshPath = entry.path().c_str();
#endif
                auto mesh = loadOBJ(meshPath);
                MeshRender render;
                render.load_mesh(mesh.vertices, mesh.triangles, mesh.vertex_texcoords, mesh.tex_triangles);
                ModalSound modal;
                modal.link_mesh_render(&render);

                float currMinScale = 1e20;
                for(int i = 0; i < 7; i++)
                {
                    auto eigenPath = std::string(ASSET_DIR) + std::string("/eigen/") + modelName + "_" + MaterialConst::names[i] + std::string(".npz");
                    auto ffatPath = std::string(ASSET_DIR) + std::string("/acousticMap/") + modelName + "_" + MaterialConst::names[i] + std::string(".npz");
                    auto voxelPath = std::string(ASSET_DIR) + std::string("/voxel/") + modelName + std::string(".npy");
                    modal.SetModal(eigenPath.c_str(), ffatPath.c_str(), voxelPath.c_str());
                    modal.SetMaterial(i, false);
                    modal.AdjustSoundScale();
                    if(Correction::soundScale < currMinScale)
                        currMinScale = Correction::soundScale;
                } 
                Correction::allSoundScales.emplace(modelName, currMinScale);
                correctionFile << modelName << " " << currMinScale << "\n";
            }
            ++preprocessedCnt;
            std::cerr << preprocessedCnt << " files have finished preproecessing.\r";
        }
        std::cout << "\n";
        return;
    };
}