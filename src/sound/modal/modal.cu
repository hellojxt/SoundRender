#include "hip/hip_runtime.h"
#include "modal.h"
#include <queue>

namespace SoundRender
{
    // This is only used for non-filtered data.
    void ModalSound::FilterAndFillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        using namespace MaterialConst;
        using valInfo = std::tuple<float, size_t, float>;
        struct cmp
        {
            bool operator()(const valInfo a, const valInfo b) const
            {
                return std::get<0>(a) > std::get<0>(b);
            }
        };

        float *valueData = rawEigenValues.data<float>();
        size_t valueNum = rawEigenValues.num_vals;

        std::vector<valInfo> fitInfos;
        fitInfos.reserve(valueNum);

        int cnt = 0;
        for (size_t i = 0; i < valueNum; i++)
        {
            float lambda = valueData[i];
            float omega = std::sqrt(lambda);
            float ksi = (alpha + beta * lambda) / (2 * omega);
            float frequency = omega * std::sqrt(1 - ksi * ksi) / 2 * PI;
            if (frequency < 20 || frequency > 20000) // can not be heard.
                continue;
            ++cnt;
            fitInfos.emplace_back(frequency, i, lambda);
        }
        // TODO : or, we can use nth_element and traverse the vec.
        // cmp CmpStd;
        // auto ele = *std::nth_element(fitInfos.begin(), fitInfos.begin() + cnt, selectNum, CmpStd);
        // for(size_t i = 0; i < cnt; i++)
        // {
        //      if(cmpStd(fitsInfo[i], ele))
        //      {
        //          int index = std::get<1>(val);
        //          float lambda = std::get<2>(val);
        //          modalInfos.emplace_back(lambda, index, rawEigenVecs);
        //      }
        // }
        // modalInfos.emplace_back(std::get<2>(ele), std::get<1>(ele), rawEigenVecs);
        std::priority_queue<valInfo, std::vector<valInfo>, cmp> infoHeap(fitInfos.begin(), fitInfos.begin() + cnt);
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        for (int i = 0; i < selectNum; i++)
        {
            const valInfo &val = infoHeap.top();
            size_t index = std::get<1>(val);
            float lambda = std::get<2>(val);
            modalInfos.emplace_back(lambda, index, rawEigenVecs, rawFFAT);
            infoHeap.pop();
        }
        return;
    }

    // This one is used for filtered data.
    void ModalSound::FillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        float *valueData = rawEigenValues.data<float>();
        for (int i = 0; i < selectNum; i++)
        {
            modalInfos.emplace_back(valueData[i], i, rawEigenVecs, rawFFAT);
        }
        return;
    }

    void ModalSound::FillVertID(cnpy::NpyArray &rawVoxelData)
    {
        int *voxelData = rawVoxelData.data<int>();
        size_t xSize = rawVoxelData.shape[0], ySize = rawVoxelData.shape[1],
               zSize = rawVoxelData.shape[2];

        // allocate memory.
        vertData.resize(xSize + 1);
        for (auto &i : vertData)
        {
            i.resize(ySize + 1);
            for (auto &j : i)
            {
                j.resize(zSize + 1, 0);
            }
        }

        // check voxel in object.
        for (size_t i = 0; i < xSize; i++)
        {
            size_t baseIndex1 = i * ySize * zSize;
            for (size_t j = 0; j < ySize; j++)
            {
                size_t baseIndex = j * zSize + baseIndex1;
                for (size_t k = 0; k < zSize; k++)
                {
                    if (voxelData[baseIndex + k] == 1)
                    {
                        for (auto &offset : MaterialConst::offsets)
                        {
                            vertData[i + offset[0]][j + offset[1]][k + offset[2]] = 1;
                        }
                    }
                }
            }
        }

        // put id in vert bucket.
        int cnt = 0;
        for (auto &i : vertData)
        {
            for (auto &j : i)
            {
                for (int &k : j)
                {
                    if (k == 1)
                    {
                        k = cnt;
                        ++cnt;
                    }
                }
            }
        }
        return;
    }

    float Lerp(float x1, float x2, float coeff)
    {
        return x1 * coeff + x2 * (1 - coeff);
    }

}