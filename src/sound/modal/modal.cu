#include "hip/hip_runtime.h"
#include "modal.h"
#include <queue>

namespace SoundRender
{
    // This is only used for non-filtered data.
    void ModalSound::FilterAndFillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        using namespace MaterialConst;
        using valInfo = std::tuple<float, size_t, float>;
        struct cmp
        {
            bool operator()(const valInfo a, const valInfo b) const
            {
                return std::get<0>(a) > std::get<0>(b);
            }
        };

        float *valueData = rawEigenValues.data<float>();
        size_t valueNum = rawEigenValues.num_vals;

        std::vector<valInfo> fitInfos;
        fitInfos.reserve(valueNum);

        int cnt = 0;
        for (size_t i = 0; i < valueNum; i++)
        {
            float lambda = valueData[i];
            float omega = std::sqrt(lambda);
            float ksi = (alpha + beta * lambda) / (2 * omega);
            float frequency = omega * std::sqrt(1 - ksi * ksi) / 2 * PI;
            if (frequency < 20 || frequency > 20000) // can not be heard.
                continue;
            ++cnt;
            fitInfos.emplace_back(frequency, i, lambda);
        }
        // TODO : or, we can use nth_element and traverse the vec.
        // cmp CmpStd;
        // auto ele = *std::nth_element(fitInfos.begin(), fitInfos.begin() + cnt, selectNum, CmpStd);
        // for(size_t i = 0; i < cnt; i++)
        // {
        //      if(cmpStd(fitsInfo[i], ele))
        //      {
        //          int index = std::get<1>(val);
        //          float lambda = std::get<2>(val);
        //          modalInfos.emplace_back(lambda, index, rawEigenVecs);
        //      }
        // }
        // modalInfos.emplace_back(std::get<2>(ele), std::get<1>(ele), rawEigenVecs);
        std::priority_queue<valInfo, std::vector<valInfo>, cmp> infoHeap(fitInfos.begin(), fitInfos.begin() + cnt);
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        for (int i = 0; i < selectNum; i++)
        {
            const valInfo &val = infoHeap.top();
            size_t index = std::get<1>(val);
            float lambda = std::get<2>(val);
            modalInfos.emplace_back(lambda, index, rawEigenVecs, rawFFAT);
            infoHeap.pop();
        }
        return;
    }

    // This one is used for filtered data.
    void ModalSound::FillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        float *valueData = rawEigenValues.data<float>();
        for (int i = 0; i < selectNum; i++)
        {
            modalInfos.emplace_back(valueData[i], i, rawEigenVecs, rawFFAT);
        }
        return;
    }

    void ModalSound::FillVertID(cnpy::NpyArray &rawVoxelData)
    {
        int *voxelData = rawVoxelData.data<int>();
        size_t xSize = rawVoxelData.shape[0], ySize = rawVoxelData.shape[1],
               zSize = rawVoxelData.shape[2];

        // allocate memory.
        vertData.resize(xSize + 1);
        for (auto &i : vertData)
        {
            i.resize(ySize + 1);
            for (auto &j : i)
            {
                j.resize(zSize + 1, 0);
            }
        }

        // check voxel in object.
        for (size_t i = 0; i < xSize; i++)
        {
            size_t baseIndex1 = i * ySize * zSize;
            for (size_t j = 0; j < ySize; j++)
            {
                size_t baseIndex = j * zSize + baseIndex1;
                for (size_t k = 0; k < zSize; k++)
                {
                    if (voxelData[baseIndex + k] == 1)
                    {
                        for (auto &offset : MaterialConst::offsets)
                        {
                            vertData[i + offset[0]][j + offset[1]][k + offset[2]] = 1;
                        }
                    }
                }
            }
        }

        // put id in vert bucket.
        int cnt = 0;
        for (auto &i : vertData)
        {
            for (auto &j : i)
            {
                for (int &k : j)
                {
                    if (k == 1)
                    {
                        k = cnt;
                        ++cnt;
                    }
                }
            }
        }
        return;
    }

    inline float Lerp(float x1, float x2, float coeff)
    {
        return x1 * coeff + x2 * (1 - coeff);
    }

    std::pair<float, float>  ModalSound::GetModalResult(ModalInfo &modalInfo)
    {
        const float camx = mesh_render->camera.Position[0], 
            camy = mesh_render->camera.Position[1],
            camz = mesh_render->camera.Position[2];
        const float r = std::sqrt(camx * camx + camy * camy + camz * camz);
        const size_t ffatColNum = modalInfo.ffat[0].size();
        // Here we need row and col sample intervals are the same, otherwise changes are needed.
        const float sampleIntervalRep = ffatColNum / PI;

        float theta = std::acos(camz / r);
        float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= -1e-5f && camy >= -1e-5f ? 0.0f : std::atan2(camy, camx) + PI;

        float colInter = theta * sampleIntervalRep, rowInter = phi * sampleIntervalRep;
        int col = static_cast<int>(colInter);
        float colFrac = colInter - static_cast<float>(col);
        int row = static_cast<int>(rowInter);
        float rowFrac = rowInter - static_cast<float>(row);
        // printf("ffatSize : %zu * %zu; row = %d, col = %d\n", modalInfo.ffat.size(), ffatColNum, row, col);
        
        // bi-Lerp.
        // float interResult = Lerp(Lerp(modalInfo.ffat[row][col], modalInfo.ffat[row + 1][col], rowFrac),
        //                          Lerp(modalInfo.ffat[row][col + 1], modalInfo.ffat[row + 1][col + 1], rowFrac), colFrac);

        float interResult = 1.0f;

        float p = interResult / r;
        float q = modalInfo.coeff1 * modalInfo.q1 + modalInfo.coeff2 * modalInfo.q2 + modalInfo.coeff3 * modalInfo.f;
        modalInfo.q2 = modalInfo.q1, modalInfo.q1 = q; // update q.
        modalInfo.f = 0;

        // printf("camx = %f, camy = %f, camz = %f, ffatColNum = %zu, theta = %f, phi = %f, colInter = %f, col = %d, rowInter = %f,"
        // "row = %d, modalInfo.ffat : [row][col] = %f, [row+1][col] = %f, [row][col+1]=%f, [row+1][col+1]=%f, interResult = %f\n", 
        // camx, camy, camz, ffatColNum, theta, phi, colInter, col, rowInter, row, modalInfo.ffat[row][col], modalInfo.ffat[row+1][col],
        // modalInfo.ffat[row][col+1],modalInfo.ffat[row+1][col+1],interResult);

        return {p, q};
    };
}