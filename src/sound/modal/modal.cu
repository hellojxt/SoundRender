#include "hip/hip_runtime.h"
#include "modal.h"
#include <queue>
#include <unordered_map>
#include <mutex>
#include <algorithm>
namespace SoundRender
{
    float3 GetTriangleCenter(int3 tri, CArr<float3> &vertArr)
    {
        return (vertArr[tri.x] + vertArr[tri.y] + vertArr[tri.z]) / 3.0f;
    }

    float3 GetTriangleNormal(int3 tri, CArr<float3> &vertArr)
    {
        float3 e1 = vertArr[tri.x] - vertArr[tri.y], e2 = vertArr[tri.y] - vertArr[tri.z];
        return normalize(cross(e1, e2));
    }

    ModalInfo::ModalInfo(float lambda, size_t index, cnpy::NpyArray &eigenVecs, cnpy::NpyArray &ffats)
    {
        using namespace MaterialConst;
        float omega = std::sqrt(lambda);
        float ksi = (alpha + beta * lambda) / (2 * omega);
        float omega_prime = omega * std::sqrt(1 - ksi * ksi);
        float epsilon = std::exp(-ksi * omega * timestep);
        float sqrEpsilon = epsilon * epsilon;
        float theta = omega_prime * timestep;
        float gamma = std::asin(ksi);

        coeff1 = 2 * epsilon * std::cos(theta);
        coeff2 = -sqrEpsilon;

        float coeff3_item1 = epsilon * std::cos(theta + gamma);
        float coeff3_item2 = sqrEpsilon * std::cos(2 * theta + gamma);
        coeff3 = 2 * (coeff3_item1 - coeff3_item2) / (3 * omega * omega_prime);

        size_t rank = eigenVecs.shape[0], colNum = eigenVecs.shape[1];
        eigenVec.reserve(rank);
        float *eigenVecsData = eigenVecs.data<float>();
        for (size_t i = 0; i < rank; i++)
        {
            eigenVec.push_back(eigenVecsData[i * colNum + index]);
        }

        size_t ffatRowNum = ffats.shape[1], ffatColNum = ffats.shape[2];
        double *ffatsData = ffats.data<double>() + ffatRowNum * ffatColNum * index;
        ffat.reserve(ffatRowNum);
        for (size_t i = 0; i < ffatRowNum; i++)
        {
            size_t baseIndex = i * ffatColNum;
            ffat.emplace_back(ffatsData + baseIndex, ffatsData + baseIndex + ffatColNum);
        }
        f = q1 = q2 = 0;
    }

    void ModalSound::init(const char *eigenPath, const char *ffatPath, const char *voxelPath)
    {
        cnpy::npz_t eigenData = cnpy::npz_load(eigenPath);
        cnpy::NpyArray &rawEigenValues = eigenData["vals"];                 // get S
        cnpy::NpyArray &rawEigenVecs = eigenData["vecs"];                   // get U
        cnpy::NpyArray rawFFAT = cnpy::npz_load(ffatPath, "feats_out_far"); // get FFAT.
        assert(rawFFAT.word_size == sizeof(double));
        assert(rawEigenValues.word_size == sizeof(float));
        assert(rawEigenVecs.word_size == sizeof(float));

        FillModalInfos(rawEigenValues, rawEigenVecs, rawFFAT);
        cnpy::NpyArray rawVoxelData = cnpy::npy_load(voxelPath);
        assert(rawVoxelData.word_size == sizeof(int));
        FillVertID(rawVoxelData);
        select_voxel_idx = make_int3(-1, -1, -1);
        select_point = make_float3(-1, -1, -1);
        force = 0.5;
        for (int i = 0; i < 8; i++)
        {
            select_voxel_vertex_idx[i] = -1;
        }
    }

    void ModalSound::update()
    {
        ImGui::Text("Here is ModalSound Module");
        ImGui::Text("Mesh has %d vertices and %d triangles", mesh_render->vertices.size(), mesh_render->triangles.size());
        ImGui::Text("Camera position:  (%f, %f, %f)", mesh_render->camera.Position.x, mesh_render->camera.Position.y, mesh_render->camera.Position.z);
        ImGui::SliderFloat("Click Force", &force, 0.0f, 1.0f);
        ImGui::Text("Force: %f", force);
        ImGui::Text("Selected Triangle Index: %d", mesh_render->selectedTriangle);
        reset_modal_f();
        if (mesh_render->soundNeedsUpdate)
        {
            int3 tri = mesh_render->triangles[mesh_render->selectedTriangle];
            select_point = GetTriangleCenter(tri, mesh_render->vertices);
            select_voxel_idx = GetNormalizedID(select_point);
            auto norm = GetTriangleNormal(tri, mesh_render->vertices);
            for (int i = 0; i < 8; i++)
            {
                auto offset = MaterialConst::offsets[i];
                auto id = vertData[select_voxel_idx.x + offset[0]][select_voxel_idx.y + offset[1]][select_voxel_idx.z + offset[2]] * 3;
                select_voxel_vertex_idx[i] = id;
                for (auto &modalInfo : modalInfos)
                {
                    auto mode_f = modalInfo.eigenVec[id] * norm.x + modalInfo.eigenVec[id + 1] * norm.y + modalInfo.eigenVec[id + 2] * norm.z;
                    modalInfo.f += mode_f * force / 8;
                }
            }
            mesh_render->soundNeedsUpdate = false;
        }
        ImGui::Text("Selected Triangle Center: (%f, %f, %f)", select_point.x, select_point.y, select_point.z);
        ImGui::Text("Selected Voxel Index: (%d, %d, %d)", select_voxel_idx.x, select_voxel_idx.y, select_voxel_idx.z);
        ImGui::Text("Selected Voxel Vertex Index: ");
        for (int i = 0; i < 8; i++)
        {
            ImGui::Text("%d", select_voxel_vertex_idx[i]);
        }
    }

    int3 ModalSound::GetNormalizedID(float3 center)
    {
        size_t voxelNum = vertData.size() - 1;
        float3 bbMin = mesh_render->bbox_min, bbMax = mesh_render->bbox_max;
        float3 relative_coord = (center - bbMin) / (bbMax - bbMin);
        return make_int3((relative_coord * (float)voxelNum));
    }

    void ModalSound::FillModalInfos(cnpy::NpyArray &rawEigenValues, cnpy::NpyArray &rawEigenVecs, cnpy::NpyArray &rawFFAT)
    {
        size_t selectNum = rawFFAT.shape[0];
        modalInfos.reserve(selectNum);
        float *valueData = rawEigenValues.data<float>();
        for (int i = 0; i < selectNum; i++)
        {
            modalInfos.emplace_back(valueData[i], i, rawEigenVecs, rawFFAT);
        }
        return;
    }

    void ModalSound::FillVertID(cnpy::NpyArray &rawVoxelData)
    {
        int *voxelData = rawVoxelData.data<int>();
        size_t xSize = rawVoxelData.shape[0], ySize = rawVoxelData.shape[1],
               zSize = rawVoxelData.shape[2];

        // allocate memory.
        vertData.resize(xSize + 1);
        for (auto &i : vertData)
        {
            i.resize(ySize + 1);
            for (auto &j : i)
            {
                j.resize(zSize + 1, 0);
            }
        }

        // check voxel in object.
        for (size_t i = 0; i < xSize; i++)
        {
            size_t baseIndex1 = i * ySize * zSize;
            for (size_t j = 0; j < ySize; j++)
            {
                size_t baseIndex = j * zSize + baseIndex1;
                for (size_t k = 0; k < zSize; k++)
                {
                    if (voxelData[baseIndex + k] == 1)
                    {
                        for (auto &offset : MaterialConst::offsets)
                        {
                            vertData[i + offset[0]][j + offset[1]][k + offset[2]] = 1;
                        }
                    }
                }
            }
        }

        // put id in vert bucket.
        int cnt = 0;
        for (auto &i : vertData)
        {
            for (auto &j : i)
            {
                for (int &k : j)
                {
                    if (k == 1)
                    {
                        k = cnt;
                        ++cnt;
                    }
                }
            }
        }
        return;
    }

    inline float Lerp(float x1, float x2, float coeff)
    {
        return x1 * coeff + x2 * (1 - coeff);
    }

    float ModalSound::GetFFATFactor(ModalInfo& modalInfo)
    {
        const float camx = mesh_render->camera.Position[0],
            camy = mesh_render->camera.Position[1],
            camz = mesh_render->camera.Position[2];
        const float r = std::sqrt(camx * camx + camy * camy + camz * camz) + 1e-4f; // to prevent singular point.
        const size_t ffatRowNum = modalInfo.ffat.size();
        const size_t ffatColNum = modalInfo.ffat[0].size();
        // Here we need row and col sample intervals are the same, otherwise changes are needed.
        const float rowSampleIntervalRep = ffatRowNum / (2 * PI);
        const float colSampleIntervalRep = ffatColNum / PI;

        float theta = std::acos(camz / r);
        float phi = camy <= 1e-5f && camx <= 1e-5f && camx >= -1e-5f && camy >= -1e-5f ? 0.0f : std::fmod(std::atan2(camy, camx) + 2 * PI, 2* PI);

        float colInter = theta * colSampleIntervalRep, rowInter = phi * rowSampleIntervalRep;
        int col = static_cast<int>(colInter);
        float colFrac = colInter - static_cast<float>(col);
        int row = static_cast<int>(rowInter);
        float rowFrac = rowInter - static_cast<float>(row);
        // bi-Lerp.
        int nextRow = (row + 1) % ffatRowNum, nextCol = (col + 1) % ffatColNum;
        float interResult = Lerp(Lerp(modalInfo.ffat[row][col], modalInfo.ffat[nextRow][col], rowFrac),
                                 Lerp(modalInfo.ffat[row][nextCol], modalInfo.ffat[nextRow][nextCol], rowFrac), colFrac);

        // printf("camx = %f, camy = %f, camz = %f, ffatColNum = %zu, theta = %f, phi = %f, colInter = %f, col = %d, rowInter = %f,"
        // "row = %d, modalInfo.ffat : [row][col] = %f, [row+1][col] = %f, [row][col+1]=%f, [row+1][col+1]=%f, interResult = %f\n",
        // camx, camy, camz, ffatColNum, theta, phi, colInter, col, rowInter, row, modalInfo.ffat[row][col], modalInfo.ffat[row+1][col],
        // modalInfo.ffat[row][col+1],modalInfo.ffat[row+1][col+1], interResult);

        return interResult / r;
    }
}